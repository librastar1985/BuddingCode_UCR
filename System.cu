#include "hip/hip_runtime.h"
#include "System.h"
#include "SystemStructures.h" 
#include "AreaTriangles.h"
#include "BendingTriangles.h"
#include "MemRepulsionSprings_universal.h"
#include "MemRepulsionSprings_local.h"
#include "MemRepulsionEnergy.h"
#include "LinearSprings.h"
#include "LJSprings.h"
#include "LJSprings_LJ.h"
#include "NodeAdvance.h"
#include "BucketScheme.h"//Currently not inused, will have to revisit if modeling a large system. 
#include "Storage.h" 
#include "Edgeswap_test.h"
#include "SystemBuilder.h"
#include <vector>
#include "VolumeComp.h"
#include "VolumeSprings.h"
#include <bits/stdc++.h>
#include "LineTensionSprings.h"
#include <math.h>
#include "TurgorForce.h"
// #include "LJSprings.h"
// #include "LJSprings_LJ.h"

int count_bigger(const std::vector<int>& elems) {
    return std::count_if(elems.begin(), elems.end(), [](int c){return c >= 0;});
}

System::System() {};

void System::Solve_Forces(){

	thrust::fill(coordInfoVecs.nodeForceX.begin(), coordInfoVecs.nodeForceX.end(), 0.0);
	thrust::fill(coordInfoVecs.nodeForceY.begin(), coordInfoVecs.nodeForceY.end(), 0.0);
	thrust::fill(coordInfoVecs.nodeForceZ.begin(), coordInfoVecs.nodeForceZ.end(), 0.0);
	
	//setBucketScheme();
	ComputeLinearSprings(
		generalParams, 
		coordInfoVecs,
		linearSpringInfoVecs, 
		ljInfoVecs);
	
	ComputeAreaTriangleSprings(
		
		generalParams,
		coordInfoVecs,
		areaTriangleInfoVecs);
	
	ComputeTurgorSprings(
		generalParams,
		coordInfoVecs,
		areaTriangleInfoVecs
	);
	
	ComputeCosTriangleSprings(
		generalParams,
		coordInfoVecs,  
		bendingTriangleInfoVecs); 
	
	// ComputeMemRepulsionSprings_universal(
	// 	coordInfoVecs,
	// 	linearSpringInfoVecs, 
	// 	capsidInfoVecs,
	// 	generalParams,
	// 	auxVecs); //Universal volume exlcusion purpose. Slower than local volume exclusion but use this one if you anticipate distant nodes
	              //to get close to each other.

	ComputeMemRepulsionSprings_local(
		coordInfoVecs,
		linearSpringInfoVecs, 
		capsidInfoVecs,
		generalParams,
		auxVecs); //Local volume exlcusion purpose. This one only calculates a number of neighboring nodes surrounding the target node.
				  //Only use this if you know the system won't have distant nodes getting too close.

	ComputeVolume(
		generalParams,
		coordInfoVecs,
		linearSpringInfoVecs,
		ljInfoVecs); //Compute volume of the cell. Not sure what would happen if we don't have a closed system (ex. a piece of membrane)

	
	/*ComputeVolumeSprings(
		coordInfoVecs,
		linearSpringInfoVecs, 
		capsidInfoVecs,
		generalParams,
		auxVecs);*/ //For volume constraint purpose.

	/* if (generalParams.true_current_total_volume/initial_volume >= 1.25){
	ComputeLineTensionSprings(
		generalParams,
		coordInfoVecs,
		linearSpringInfoVecs);
	} */ //Line tension related calculation is currently separated from this general force computation function.
		
};


void System::solveSystem() {
	// double portion_of_bud_for_wall_insertion = 0.25;
	// double ratio_for_restricted_wall_insertion = 3.5;
	// std::cout<<"Only the top "<<portion_of_bud_for_wall_insertion<<" of the budding region are suitable for cell wall insertion,"<<std::endl;
	// std::cout<<"where the restriction occurs when the the distance between tip of the bud and the septin ring is "<<ratio_for_restricted_wall_insertion<<std::endl;
	// std::cout<<"times larger than the distance at the initial condition"<<std::endl;
	
	generalParams.nodeMass = 1.0;
	int GROWTH_COUNTER = 0;
	int min_num_edge_loop = 1;
	std::cout<<"min_num_edge_loop for edgeswap = "<<min_num_edge_loop<<std::endl;

	// std::random_device rand_dev;
	// // std::mt19937 generator2(rand_dev());
	// std::mt19937 generator_edgeswap(rand_dev());

	double MAX_VOLUME_RATIO = 2.0;
	double MAX_BUD_AREA_RATIO = 100.0;
	int MAX_GROWTH_NUMBER = 1;
	std::cout<<"MAX_GROWTH_NUMBER (# of edge to expand) = "<<MAX_GROWTH_NUMBER<<std::endl;
	int GROWTH_FREQUENCY = 90;//25;//95;//70;//25*3;
	std::cout<<"GROWTH_FREQ (how many times Max_Runtime has to be reached to perform growth"<<GROWTH_FREQUENCY<<std::endl;
	double energy_gradient_threshold = 0.02;//0.01;
	std::cout<<"ENERGY_GRADIENT_THRESHOLD = "<<energy_gradient_threshold<<std::endl;

	// generalParams.kT_growth = 1.0;
	generalParams.SCALE_TYPE = 3; 
	// 0:= Gaussian-like weakening
	// 1:= a1*(pow(x,b)) + a2*(1-pow(x,b)) type weakening
	// 2:= pure Gaussian weakening
	// 3:= isotropic
	// 4:= hill equation
	//Note that (3) is used in combination with sigma = INT_MAX;

	std::cout<<"SCALE TYPE = "<<generalParams.SCALE_TYPE<<std::endl;
	std::cout<<"0:= sigmoidal Gaussian-like weakening, 1:= a1*(pow(x,b)) + a2*(1-pow(x,b)) type weakening, 2:= pure Gaussian weakening, 3:= isotropic, 4:= hill equation"<<std::endl;
	
	if (SCALE_TYPE == 0){
		generalParams.gausssigma = 0.1;
		std::cout<<"gausssigma (this is only meaningful for the SCALE_TYPE = 0 case) = "<<generalParams.gausssigma<<std::endl;
	}
	if (SCALE_TYPE == 1){
		generalParams.scaling_pow = 2.0;
		std::cout<<"scaling_pow (this is only meaningful for SCALE_TYPE = 1 case) = "<<generalParams.scaling_pow<<std::endl;
	}
	if (SCALE_TYPE == 2){
		double sigma = INT_MAX; 
		double sigma_true = sqrt(0.5); //This is the variance used to calculate the scaling of the wall weakening.
		std::cout<<"initial sigma (for gradient distribution variance), based on initial distribution of Cdc42, if using true gaussian weakening = "<<sigma<<std::endl;
		std::cout<<"If sigma = INT_MAX, then we have isotropic weakening scenario"<<std::endl;
		std::cout<<"true sigma (for gaussian-related distribution variance) = "<<sigma_true<<std::endl;
	}

	generalParams.strain_threshold = 0.05;//0.01; //The threshold value for avg area strain to trigger cell wall insertion
	std::cout<<"GROWTH: critical strain threshold used for insertion probability calculation = "<<generalParams.strain_threshold<<std::endl;
	generalParams.safeguardthreshold = 9;
	std::cout<<"NEIGHBOR SAFE GUARD THRESHOLD = "<<generalParams.safeguardthreshold<<std::endl;
	//safeguardthreshold is the maximum number of neighboring nodes a node can have.

	
	generalParams.insertion_energy_cost = -log(0.0025);//For probablistic cell surface insertion, currently not in use
	// std::cout<<"GROWTH: material insertion energy cost (dependent on local chemical concentration) = "<<generalParams.insertion_energy_cost<<std::endl;
	
	generalParams.growth_energy_scaling = 1.0;//0.01375;//For probablistic cell surface insertion, currently not in use
	// std::cout<<"GROWTH ENERGY SCALING FOR MATERIAL INSERTION PROBABILITY = "<<generalParams.growth_energy_scaling<<std::endl;
	
	std::vector<int> nodes_in_growth;//for cases where we need a designed growth zone (growth via edge extension) 
	std::vector<int> triangles_in_growth;//for cases where we need a designed growth zone (growth via edge extension) 
	std::vector<int> edges_in_growth;//for cases where we need a designed growth zone (growth via edge extension) 

	double dtb; //dtb := distance to boundary, used in SCALE_TYPE = 4 type simulation
	double dtb_max; //dtb_max := the max distance used to calculate the distance ratio in the Hill equation.

	//////////////////////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////////////
	////////////////////////// PARAMETER SETTINGS ////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////////////

	auto edgeswap_ptr = std::make_shared<Edgeswap>(coordInfoVecs, generalParams);//Pointer that links this .cu file to Edgeswap_test.cpp
	auto build_ptr = weak_bld_ptr.lock();//upgrade weak builder to access host variables.

	///// relaxation related /////
	bool runSim = true; //This tells the program that we intend to run the simulation. It is also used as a stopping criteria for thhe simulation.
	double Max_Runtime = generalParams.dt*50;//Max number of steps to run in one relaxation loop.
	double Max_RunStep = Max_Runtime/generalParams.dt;
	std::cout<<"Max runtime = "<<Max_Runtime<<std::endl;
	std::cout<<"Max runstep = "<<Max_RunStep<<std::endl;
	double minimal_run_time_ratio = 1.0;//To force a number of relaxation steps to run regardless if the energy gradient (or difference) is already below the threshold.
	int RECORD_TIME = 1;//round(Max_RunStep/2);
	std::cout<<"Record frequency = "<<RECORD_TIME<<std::endl;
	int translate_frequency = 10;
	std::cout<<"recentering of the model cell frequency = "<<translate_frequency<<std::endl;
	double old_total_energy = 0.0;
	double new_total_energy = 0.0;
	double energy_gradient = 0.0;
	double energy_rep = 0.0;
	int Num_of_step_run = 0;
	double min_energy;
	//////////////////////////////

	///// edge-swap related //////
	int num_edge_loop; //The number of edges tested for edge-swap algorithhm.
	double initial_kT;
	initial_kT = generalParams.kT;//This is to record the initial kT value we use for the edge-swap algorithm.
	double SAMPLE_SIZE = 0.05;//The percentage of total number of edge tested for edge-swap.
	std::cout<<"Sample ratio: "<<SAMPLE_SIZE<<std::endl;
	std::cout<<"If the Sample raio is 0, it means we have chosen a fixed number of attempt throughout the simulation"<<std::endl;
	//This determines the number of edges to test for bondflip remeshing
	//////////////////////////////

	///// growth related /////////
	int TOTAL_GROWTH_COUNTER = 0;//For growth purpose, see later section of the code for uses.
	int TOTAL_GROWTH_ATTEMPT = 0;//For growth purpose, see later section of the code for uses.
	//int GROWTH_TIME = 1;
	//std::cout<<"Growth frequency = "<<GROWTH_TIME<<std::endl;
	int NUMBER_OF_GROWTH_EVENT = 1000*2;
	std::cout<<"Number of maximally allowed growth event = "<<NUMBER_OF_GROWTH_EVENT<<" which used to terminate the simulation if not enough growth is encountered for a prolonged simulation."<<std::endl;
	int NUMBER_OF_TARGETED_GROWTH_EVENT = 1000;
	int GROWTH_FREQUENCY_SCALE = 4;
	std::cout<<"GROWTH FREQ SCALE: decides how many growth algorithm must be triggered before recording the result"<<std::endl;
	//////////////////////////////
	
	int NKBT = GROWTH_FREQUENCY*NUMBER_OF_GROWTH_EVENT;//10000;//7500;
	std::cout<<"Number of edge-swap per kBT value (or total number of edge-swap if kBT is fixed (so not a simulated annealing process)) = "<<NKBT<<std::endl;
	
	double min_kT = -0.1;//0.21;
	std::cout<<"min kT for simulation termination = "<<min_kT<<std::endl;
	
	//std::cout<<"initial LJ-x : "<< ljInfoVecs.LJ_PosX <<std::endl;
	//std::cout<<"initial LJ-y : "<< ljInfoVecs.LJ_PosY <<std::endl;
	//std::cout<<"initial LJ-z : "<< ljInfoVecs.LJ_PosZ <<std::endl;
    
	generalParams.true_num_edges = 0;
	for (int i = 0; i < coordInfoVecs.num_edges; i++){
		if (coordInfoVecs.edges2Nodes_1[i] != INT_MAX && coordInfoVecs.edges2Nodes_2[i] != INT_MAX){
			generalParams.true_num_edges += 1;
		}
	}//This loop is important because it calculates the actual number of edges in the system (since we do not explicitly delete edge info)
	 //This will be used everytime we want to print out VTK files. If not used, the VTK file cannot be visualized.

	/////////////////////////////////////////////////////////////////
	/////////////////////// MEMBRANE RELATED ////////////////////////
	/////////////////////////////////////////////////////////////////
	
	std::vector<double> nodenormal_1(generalParams.maxNodeCount, 0.0);
	std::vector<double> nodenormal_2(generalParams.maxNodeCount, 0.0);
	std::vector<double> nodenormal_3(generalParams.maxNodeCount, 0.0);
	int reduce_counter = 0;

	double VOLUME_FACTOR = MAX_VOLUME_RATIO;//1.6;//2.25; //VOLUME_FACTOR determines the target volume which equals to VOLUME_FACTOR*initial_volume.
	//double tip_depth = 0.5;//tip_depth is currently unused.

	double LINE_TENSION_THRESHOLD = -10000.0; //The factor volume of the cell must increase before line tension becomes active.
	std::cout<<"LINE TENSION THRESHOLD for activation of line tension = "<<LINE_TENSION_THRESHOLD<<std::endl;
	double VOLUME_THRESHOLD = 0.0; //The factor volume of the cell must increase before surface weakening becomes active.//Currently not in use.
	// std::cout<<"VOLUME THRESHOLD for activation of weakened membrane = "<<VOLUME_THRESHOLD<<std::endl;//Currently not in use.
	
	double weakened = 1.90;//6.0;
	//'weakened' determines the minimum height of the z-coordinate of the membrane node to be considered in the area of weakened mechanical properties.//Currently not in use.
	//double tip_base = 6.0;//tip_base currently unused.

	//Calculating center of the cell
	for (int i = 0; i < generalParams.maxNodeCount; i++){
		generalParams.centerX += coordInfoVecs.nodeLocX[i];
		generalParams.centerY += coordInfoVecs.nodeLocY[i];
		generalParams.centerZ += coordInfoVecs.nodeLocZ[i];
	}
	generalParams.centerX = generalParams.centerX/generalParams.maxNodeCount;
	generalParams.centerY = generalParams.centerY/generalParams.maxNodeCount;
	generalParams.centerZ = generalParams.centerZ/generalParams.maxNodeCount;
	double displacementX, displacementY, displacementZ;
	double newcenterX, newcenterY, newcenterZ;
	//centerX, centerY, centerZ are used as the referenced origin for recentering of the mesh.

	std::vector<int> VectorShuffleForGrowthLoop; //Initialize vector to store growth candidate edges.
	// std::vector<int> VectorShuffleForFilamentLoop; //Initialize vector to store filament candidate edges.//Currently not in use.
	std::vector<int> VectorShuffleForEdgeswapLoop; //Initialize vector to store edge-swap candidate edges.

	double max_height = coordInfoVecs.nodeLocZ[35]; //NOTE: this only works for specific mesh. Use the commented out part below if you 
													//don't know the exact location of cell min and cell max indices.
	double min_height = coordInfoVecs.nodeLocZ[38];
	int max_height_index = 35;
	/*double max_height = -10000.0;
	int max_height_index = -1;
	std::vector<int> Stiffness_gradient();
    for (int k = 0; k < generalParams.maxNodeCount; k++){
        if (coordInfoVecs. nodeLocZ[k] >= max_height){
			max_height = coordInfoVecs. nodeLocZ[k];
			max_height_index = k;
            }
	}*/
	//Max and min height of the membrane nodes, these have to be changed if the mesh used is changed.

	generalParams.Rmin = 0.3012;//0.15;
	//Equilibrium length of an edge of the triangle.
	//generalParams.Rmin_growth = 0.329;//Currently not in use.

	generalParams.abs_Rmin = generalParams.Rmin;//0.15;
	std::cout<<"abs_Rmin = "<<generalParams.abs_Rmin<<std::endl;
	//Equilibrium distance between membrane node for volume exclusion.

	areaTriangleInfoVecs.initial_area = 0.039;//0.009808;//0.039;//0.03927344;//0.009817;
	std::cout<<"equilibrium triangular area = "<<areaTriangleInfoVecs.initial_area<<std::endl;
	//Equilibrium triangular area.

	ljInfoVecs.Rmin_M = 0.0;
	//Equilibrium distance between the nucleus particle and membrane.
	
	ljInfoVecs.Rcutoff_M = 0.0;
	//Maximal interaction range between the nucleus and membrane.
	
	ljInfoVecs.Rmin_LJ = 0.0;//3.0//1.0;
	//Equilibrium distance between nuclei.
	
	ljInfoVecs.Rcutoff_LJ = 0.0;//3.0;//1.0;
	//Maximal interaction range between the nuclei.
	
	ljInfoVecs.epsilon_M_att1 = 0.0;//6.0;//16.0;
	ljInfoVecs.epsilon_M_att2 = 0.0;//1.0;//1.0;
	std::cout<<"Morse_NM_D_att = "<<ljInfoVecs.epsilon_M_att1<<std::endl;
	std::cout<<"Morse_NM_a_att = "<<ljInfoVecs.epsilon_M_att2<<std::endl;
	//Coefficient for the attractive interaction between nuclei and membrane.
	
	ljInfoVecs.epsilon_M_rep1 = 0.0;//12.5;//16.0;
	ljInfoVecs.epsilon_M_rep2 = 0.0;//0.5;//1.0;
	std::cout<<"Morse_NM_D_rep = "<<ljInfoVecs.epsilon_M_rep1<<std::endl;
	std::cout<<"Morse_NM_a_rep = "<<ljInfoVecs.epsilon_M_rep2<<std::endl;
	//Coefficient for the repulsive interaction between nuclei and membrane.
	
	ljInfoVecs.epsilon_LJ_rep1 = 0.0;//10.0;//0.5;// 0.06;//7.5;
	ljInfoVecs.epsilon_LJ_rep2 = 0.0;//0.5;//1.0;//1.0;//1.0;
	std::cout<<"Morse_NN_D = "<<ljInfoVecs.epsilon_LJ_rep1<<std::endl;
	std::cout<<"Morse_NN_a = "<<ljInfoVecs.epsilon_LJ_rep2<<std::endl;
	//Coefficient of the interaction between nuclei.

	linearSpringInfoVecs.spring_constant_rep1 = 0.01;//0.023;
	linearSpringInfoVecs.spring_constant_rep2 = 9.0;//5.0;
	std::cout<<"Membrane volume exclusion Morse D = "<<linearSpringInfoVecs.spring_constant_rep1<<std::endl;
	std::cout<<"Membrane volume exclusion Morse a = "<<linearSpringInfoVecs.spring_constant_rep2<<std::endl;
	//The coefficient used for non-neighboring membrane node volume exclusion.
	//rep1 is the "D" and rep2 is the "alpha" in the standard form of Morse potential.

	generalParams.volume_spring_constant = 0.2;//(1.0/3.0)*areaTriangleInfoVecs.initial_area*1.0;
	std::cout<<"spring constant for surface normal expansion (pressure within the cell) = "<<generalParams.volume_spring_constant<<std::endl;
	
	generalParams.line_tension_constant = 0.0;//250.0;
	std::cout<<"spring constant for the septin ring (before budding) = "<<generalParams.line_tension_constant<<std::endl;
	
	generalParams.length_scale = 1.0;//0.85;//0.1577;//1.0*generalParams.Rmin;// 0.8333;
	std::cout<<"scaling factor applied to the equilibrium length of each segment of the septin ring = "<<generalParams.length_scale<<std::endl;

	// bendingTriangleInfoVecs.spring_constant = bendingTriangleInfoVecs.spring_constant*(2.0/sqrt(3));
	// This is only needed if we are working with membrane only. The (2.0/sqrt(3)) is a necessary factor to 
	// translate Helfrich continuum bending modulus to discrete bending modulus.
	// For reference: https://www.ncbi.nlm.nih.gov/pmc/articles/PMC2872218/

	double scale_linear = linearSpringInfoVecs.spring_constant*1.0;//0.25;//25.0/2.5;//75.0/15.0;
	double scale_bend = bendingTriangleInfoVecs.spring_constant*1.0;//0.05;//10.0/1.0;//75.0/7.5;
	double scale_area = areaTriangleInfoVecs.spring_constant*1.0;//0.25;//50.0/5.0;//75.0/15.0;
	std::cout<<"weakened region linear (before budding) = "<<scale_linear<<std::endl;
	std::cout<<"weakened region bend (before budding) = "<<scale_bend<<std::endl;
	std::cout<<"weakened region area (before budding) = "<<scale_area<<std::endl;
	linearSpringInfoVecs.spring_constant_weak = scale_linear;
	bendingTriangleInfoVecs.spring_constant_weak = scale_bend;
	areaTriangleInfoVecs.spring_constant_weak = scale_area;
	//Scaling of the weakend mechanical properties.

	bendingTriangleInfoVecs.initial_angle = 0.087165870975460;//0.087249;//0.04335;
	bendingTriangleInfoVecs.initial_angle_raft = 0.087165870975460;//0.087249;//0.04335;
	bendingTriangleInfoVecs.initial_angle_coat = 0.087165870975460;//0.087249;//0.04335;
	std::cout<<"equilibrium bending angle of the membrane = "<<bendingTriangleInfoVecs.initial_angle<<std::endl;
	//raft and coat are current unused due to the assumption of uniform preferred curvature.

	bendingTriangleInfoVecs.initial_angle_bud = 0.0;//bendingTriangleInfoVecs.initial_angle;///4.0;
	std::cout<<"equilibrium bending angle of the bud = "<<bendingTriangleInfoVecs.initial_angle_bud<<std::endl;
	
	/*
	// bendingTriangleInfoVecs.spring_constant_raft = 0.0;//bendingTriangleInfoVecs.spring_constant;
	// bendingTriangleInfoVecs.spring_constant_coat = 0.0;//bendingTriangleInfoVecs.spring_constant;
	// bendingTriangleInfoVecs.spring_constant = bendingTriangleInfoVecs.spring_constant*(2.0/sqrt(3));
	// bendingTriangleInfoVecs.spring_constant_raft = bendingTriangleInfoVecs.spring_constant_raft*(2.0/sqrt(3));
	// bendingTriangleInfoVecs.spring_constant_coat = bendingTriangleInfoVecs.spring_constant_coat*(2.0/sqrt(3));
	// std::cout<<"Effective bending coefficient is calculated by multiplying 2/sqrt(3)"<<std::endl;
	// std::cout<<"effective bending coefficient of the membrane = "<<bendingTriangleInfoVecs.spring_constant<<std::endl;
	// std::cout<<"effective bending coefficient of the membrane raft = "<<bendingTriangleInfoVecs.spring_constant_raft<<std::endl;
	// std::cout<<"effective bending coefficient of the membrane coat = "<<bendingTriangleInfoVecs.spring_constant_coat<<std::endl;
	*/ //This commented out section is for studying membrane with multiple subdomains with different mechanical properties

	/////////////////////////////////////////////////////////////////
	////////////////// END OF MEMBRANE RELATED //////////////////////
	/////////////////////////////////////////////////////////////////


	/////////////////////////////////////////////////////////////////
	//////////////////////// NULCEUS RELATED ////////////////////////
	/////////////////////////////////////////////////////////////////
	double beta1 = 0.0;
	double beta2 = 0.0;
	std::cout<<"manual push speed for the nucleus tip = "<<beta1<<std::endl;
	std::cout<<"manual push speed for the remainder of the nucleus = "<<beta2<<std::endl;
	//beta1 is the vertical speed (0, 0, beta1) applied to the nucleus tip.
	//beta2 is the vertical speed (0, 0, beta2) applied to the remainder of the nucleus.

	std::vector<double> V1 = {-0.0};/*, 0.0  ,  0.1966  ,  0.5547 ,  -0.4689 ,   0.2422 ,  -0.2229,
							   -0.4312 ,  -0.0185 ,   0.2887 ,   0.3187 ,   0.7140 ,  
								0.2231 ,  -0.1921 ,	  -0.5541 ,   -0.1542 ,   -0.1689 ,    0.4391 ,
							   -0.6661 ,  -0.6381 ,   0.6256 ,   0.0466 ,  -0.0610 ,   0.5134};
								*/
	std::vector<double> V2 = {0.0};/*, 0.0 ,  -0.4595 ,  -0.4129 ,   0.0954 ,   0.1764 ,   0.4186 ,
							  -0.5602 ,  -0.6082 ,  -0.5318 ,   0.3561 ,   0.0753 ,
							  -0.0917 ,  -0.2596 , 0.2871 ,  -0.3918 ,   0.5195 ,   0.5579 ,
							  -0.2805 ,   0.0133  , -0.0073 ,   0.7426 ,   0.0614 ,  -0.1506};
								*/
	std::vector<double> V3 = { 0.6390};/*, 0.0 ,  -0.5511 ,   0.0267 ,  -0.5240  , -0.4004 ,   0.2850 ,
							   0.2032 ,  -0.1771 ,   0.4048 ,   0.3461 ,  -0.2034 ,
							   0.5041 ,  -0.4535 ,	-0.1241 ,   0.5722 ,  -0.3748 ,  -0.1335 ,
							   -0.0851 ,   0.3213 ,   0.2389 ,   0.0044 ,  -0.7424 ,  -0.7450};
							   */
	//V1, V2, and V3 are the (x,y,z)-coordinate of the nucleus particles.

	for (int i = 0; i < V1.size(); i++){
		ljInfoVecs.LJ_PosX_all.push_back(V1[i]); 
		ljInfoVecs.LJ_PosY_all.push_back(V2[i]);
		ljInfoVecs.LJ_PosZ_all.push_back(V3[i]);
	}  
	
	double NUCLEUS_UPPERHEM_BASE = 0.5;
	double NUCLEUS_LOWERHEM_BASE = -0.6;
	//These values defines the z-coordinate requirement for nucleus particles to be considered tip-region or base-region. This is used to 
	// determine where to apply spring or constant force. //Currently not in use.

	/*ljInfoVecs.forceX_all.reserve(ljInfoVecs.LJ_PosX_all.size());
	ljInfoVecs.forceY_all.reserve(ljInfoVecs.LJ_PosX_all.size());
	ljInfoVecs.forceZ_all.reserve(ljInfoVecs.LJ_PosX_all.size());

	generalParams.maxNodeCountLJ = ljInfoVecs.LJ_PosX_all.size();
	std::vector<int> nucleus_in_upperhem(generalParams.maxNodeCountLJ, -1);
	std::vector<int> nucleus_in_lowerhem(generalParams.maxNodeCountLJ, -1);
	for (int i = 0; i < generalParams.maxNodeCountLJ; i++){
		if (ljInfoVecs.LJ_PosZ_all[i] > NUCLEUS_UPPERHEM_BASE){
			nucleus_in_upperhem[i] = 1;
		}
		if (ljInfoVecs.LJ_PosZ_all[i] < NUCLEUS_LOWERHEM_BASE){
			nucleus_in_lowerhem[i] = 1;
		}
	}*/
	//////////////////////////////////////////////////////////////////
	///////////////// END OF NUCLEUS RELATED /////////////////////////
	//////////////////////////////////////////////////////////////////


	//////////////////////////////////////////////////////////////////
	///////////////// Filament related ///////////////////////////////
	//////////////////////////////////////////////////////////////////
	//This part calculates the filament connecting the minimum point (in terms of z-coordinate) to the base of the nuclei cluster.

	/*std::vector<int> filament_base(generalParams.maxNodeCountLJ, -1); //= {0,1,2,3,4,5,6,7,8,9,10,11};//{35, 21, 38, etc if we need more points}
	double filament_strength = 0.0;
	double filament_strength_pull = 1.0*filament_strength;
	double filament_Rmin = ((max_height - min_height)/4.0);
	std::cout<<"filament_strength = "<<filament_strength<<std::endl;
	std::cout<<"filament_strength for vertical pull = "<<filament_strength_pull<<std::endl;
	std::cout<<"filament_Rmin = "<<filament_Rmin<<std::endl;
	
	//First, determine the initial membrane nodes having filament bridges
	//with the nuclei particles
	for (int i = 0; i < generalParams.maxNodeCountLJ; i++){
		if (i == 0){
			filament_base[i] = 35;
			continue;
		}
		for (int j = 0; j < generalParams.maxNodeCount; j++){
			double xsquared = (ljInfoVecs.LJ_PosX_all[i] - coordInfoVecs.nodeLocX[j])*
								(ljInfoVecs.LJ_PosX_all[i] - coordInfoVecs.nodeLocX[j]);
			double ysquared = (ljInfoVecs.LJ_PosY_all[i] - coordInfoVecs.nodeLocY[j])*
								(ljInfoVecs.LJ_PosY_all[i] - coordInfoVecs.nodeLocY[j]);
			double zsquared = (ljInfoVecs.LJ_PosZ_all[i] - coordInfoVecs.nodeLocZ[j])*
								(ljInfoVecs.LJ_PosZ_all[i] - coordInfoVecs.nodeLocZ[j]);
			double R = sqrt(xsquared + ysquared + zsquared);
			if (R < filament_Rmin*1.1 && j != 35){
				filament_base[i] = j;
				break;
			}
		}
	}*/
	
	//std::vector<double> filament_Rmin;
	//for (int i = 0; i < V3.size();i++){
	//	filament_Rmin.push_back(sqrt((V3[i] - coordInfoVecs.nodeLocZ[38])*(V3[i] - coordInfoVecs.nodeLocZ[38])));
	//}
	//double filament_Rmin = sqrt((V3.back() - coordInfoVecs.nodeLocZ[38])*(V3.back() - coordInfoVecs.nodeLocZ[38]));
	///////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////
	

	///////////////////////////////////////////////////////////////
	/////////// Identify region to weaken mech prop ///////////////
	/////////// Identify bdry of the system ///////////////////////
	///////////////////////////////////////////////////////////////
	std::vector<int> out;
	//int ALPHA;

	std::vector<bool> boundary_edges;
	boundary_edges.reserve(coordInfoVecs.num_edges);
	for (int i = 0; i < coordInfoVecs.num_edges; i++){
		if (coordInfoVecs.edges2Triangles_1[i] == coordInfoVecs.edges2Triangles_2[i]){
			boundary_edges.push_back(true);
		}
		else {
			boundary_edges.push_back(false);
		}
	}

	std::vector<int> edgeIndices;
	edgeIndices.reserve(coordInfoVecs.num_edges);
	for (int i = 0; i < coordInfoVecs.num_edges; ++i){
		//edgeIndices.push_back(edge_to_ljparticle[i]);
		if (boundary_edges[i] == false){
			edgeIndices.push_back(i);
		}
		else {
			edgeIndices.push_back(-1);
		}
	}

	auto it = remove_if(edgeIndices.begin(), edgeIndices.end(),  [](const int i) {return i < 0; });
	edgeIndices.erase(it, edgeIndices.end());
	
	
	//We currently hard-coded the nodes that initially belongs to the weakened region of the cell. If not, we use the 
	//commented out section marked by *****
	std::vector<int> row2 = {35 ,   76 ,   79 ,  111 ,  113 ,  151 ,  153 ,  360 ,  361 ,  362 ,  363 ,  364 ,  365 ,  505 ,  506 ,  515 ,  516 ,  593 ,  632};
	// std::vector<int> row2 = {35,76,79,111,113,151,153,360,361,362,363,364,365,505,506,515,516,593,632,840,841,842,
	//    843,844,845,1087,1090,1091,1105,1108,1109,1297,1299,1301,1309,1311,1313,1537,1539,1541,1549,1551,1553,2196,
	//   2197,2198,2199,2200,2201,2202,2203, 2204, 2205,2206,2207, 2208,2209,2210,2211,2212,2213};
	//std::vector<int> nodes_to_center;
	//generalParams.nodes_in_upperhem.resize(generalParams.maxNodeCount,-1);

	for (int i = 0; i < generalParams.maxNodeCount; i++){
		generalParams.nodes_in_upperhem[i] = -1;
	}

	for (int i = 0; i < row2.size(); i++){
		generalParams.nodes_in_upperhem[row2[i]] = 1;
	//	std::cout<<"nodes "<<i<<" "<<generalParams.nodes_in_upperhem[i]<<std::endl;		
	}
	
	// *****
	// for (int i = 0; i < generalParams.maxNodeCount; i++){
	// 	if (coordInfoVecs.nodeLocZ[i] > (generalParams.centerZ + weakened)){
	// 		generalParams.nodes_in_upperhem[i] = 1;
	// 	}
	// 	else{
	// 		generalParams.nodes_in_upperhem[i] = -1;
	// 	}
	// //	std::cout<<"nodes "<<i<<" "<<generalParams.nodes_in_upperhem[i]<<std::endl;		
	// }
	// *****

	//This is only needed if we want to deal with tip-growing cells/tissues.
	//std::vector<int> nodes_to_center;
	//std::vector<int> nodes_in_tip;
	//nodes_in_tip.resize(generalParams.maxNodeCount);
	//for (int i = 0; i < generalParams.maxNodeCount; i++){
	//	if (coordInfoVecs.nodeLocZ[i] > (generalParams.centerZ + tip_base)){
	//		nodes_in_tip[i] = 1;
	//	}
	//	else{
	//		nodes_in_tip[i] = -1;
	//	}
	//	std::cout<<"nodes "<<i<<" "<<generalParams.nodes_in_upperhem[i]<<std::endl;		
	//}

	//generalParams.triangles_in_upperhem.resize(coordInfoVecs.num_triangles);
	for (int i = 0; i < coordInfoVecs.num_triangles; i++){
		int aaa = generalParams.nodes_in_upperhem[coordInfoVecs.triangles2Nodes_1[i]];
		//std::cout<<aaa<<std::endl;
		int bbb = generalParams.nodes_in_upperhem[coordInfoVecs.triangles2Nodes_2[i]];
		//std::cout<<bbb<<std::endl;
		int ccc = generalParams.nodes_in_upperhem[coordInfoVecs.triangles2Nodes_3[i]];
		//std::cout<<ccc<<std::endl;
		if ((aaa+bbb+ccc)==3){
			generalParams.triangles_in_upperhem[i] = 1;
			//triangles_in_upperhem.push_back(i);
		}
		//else if ((aaa+bbb+ccc)==1){
		//	generalParams.triangles_in_upperhem[i] = 0;
			//triangles_in_upperhem.push_back(i);
		//}
		else{
			generalParams.triangles_in_upperhem[i] = -1;
		}
	//	std::cout<<"triangle "<<i<<" "<<generalParams.triangles_in_upperhem[i]<<std::endl;		
	}

	//std::vector<int> edges_in_upperhem;
//	generalParams.edges_in_upperhem.resize(coordInfoVecs.num_edges);
	int edges_in_upperhem_COUNT = 0;
	for (int i = 0; i < coordInfoVecs.num_edges; i++){
		int aaa = generalParams.triangles_in_upperhem[coordInfoVecs.edges2Triangles_1[i]];//generalParams.nodes_in_upperhem[coordInfoVecs.edges2Nodes_1[i]];
		int bbb = generalParams.triangles_in_upperhem[coordInfoVecs.edges2Triangles_2[i]];//generalParams.nodes_in_upperhem[coordInfoVecs.edges2Nodes_2[i]];
		if (aaa == 1 && bbb == 1){
			generalParams.edges_in_upperhem[i] = 1;
			//generalParams.edges_in_upperhem_list.push_back(i);
			generalParams.edges_in_upperhem_list[i] = i;
			edges_in_upperhem_COUNT += 1;
		}
		else if (aaa == 1 || bbb == 1){
			generalParams.edges_in_upperhem[i] = 1;
			generalParams.edges_in_upperhem_list[i] = -INT_MAX;
			edges_in_upperhem_COUNT += 1;
		}
		else{
			generalParams.edges_in_upperhem[i] = -1;
			generalParams.edges_in_upperhem_list[i] = -INT_MAX;
		}
		
	}
	std::cout<<"INITIAL EDGES IN UPPERHEM = "<<edges_in_upperhem_COUNT<<std::endl;

	int COUNTING_EDGE = 0;
	for (int y = 0; y < coordInfoVecs.num_edges; y++){
		if (generalParams.edges_in_upperhem_list[y] >= 0){
			COUNTING_EDGE += 1;
		}
		generalParams.edges_in_upperhem_list_length = COUNTING_EDGE;
	}
	

	//Find the boundary of the nodes_in_upperhem region
	//generalParams.boundaries_in_upperhem.resize(coordInfoVecs.num_edges);
	std::vector<int> boundary_node_list;
	std::vector<int> boundary_edge_list;
	for (int i = 0; i < coordInfoVecs.num_edges; i++){
		double T1 = coordInfoVecs.edges2Triangles_1[i];
		double T2 = coordInfoVecs.edges2Triangles_2[i];
		if (generalParams.triangles_in_upperhem[T1] == 1 && generalParams.triangles_in_upperhem[T2] != 1){
			generalParams.boundaries_in_upperhem[i] = 1;
			//std::cout<<generalParams.boundaries_in_upperhem[i]<<std::endl;
		//	generalParams.triangles_in_upperhem[T1] = 0;
		//	generalParams.triangles_in_upperhem[T2] = 0;
			double bdry_node1 = coordInfoVecs.edges2Nodes_1[i];
			double bdry_node2 = coordInfoVecs.edges2Nodes_2[i];
			boundary_node_list.push_back(bdry_node1);
			boundary_node_list.push_back(bdry_node2);
			boundary_edge_list.push_back(i);
			//generalParams.nodes_in_upperhem[bdry_node1] = 0;
			//generalParams.nodes_in_upperhem[bdry_node2] = 0;
			coordInfoVecs.isNodeFixed[bdry_node1] = true;
			coordInfoVecs.isNodeFixed[bdry_node2] = true;
		}
		else if (generalParams.triangles_in_upperhem[T1] != 1 && generalParams.triangles_in_upperhem[T2] == 1){
			generalParams.boundaries_in_upperhem[i] = 1;
			//std::cout<<generalParams.boundaries_in_upperhem[i]<<std::endl;
		//	generalParams.triangles_in_upperhem[T1] = 0;
		//	generalParams.triangles_in_upperhem[T2] = 0;
			double bdry_node1 = coordInfoVecs.edges2Nodes_1[i];
			double bdry_node2 = coordInfoVecs.edges2Nodes_2[i];
			boundary_node_list.push_back(bdry_node1);
			boundary_node_list.push_back(bdry_node2);
			boundary_edge_list.push_back(i);
			//generalParams.nodes_in_upperhem[bdry_node1] = 0;
			//generalParams.nodes_in_upperhem[bdry_node2] = 0;
			coordInfoVecs.isNodeFixed[bdry_node1] = true;
			coordInfoVecs.isNodeFixed[bdry_node2] = true;
		}
		else {
			generalParams.boundaries_in_upperhem[i] = -1;
			//std::cout<<generalParams.boundaries_in_upperhem[i]<<std::endl;
		}
	}
	std::cout<<"size of boundary_node_list in upperhem (this is double-counted) = "<<boundary_node_list.size()<<std::endl;
	//generalParams.eq_total_boundary_length = generalParams.boundaries_in_upperhem.size()*generalParams.Rmin;

	/*for (int i = 0; i < coordInfoVecs.num_edges; i++){
		int aaa = coordInfoVecs.edges2Nodes_1[i];
		int bbb = coordInfoVecs.edges2Nodes_2[i];
		if (aaa == 1 && bbb == 1){
			generalParams.edges_in_upperhem[i] = 1;
			generalParams.edges_in_upperhem_list.push_back(i);
		}
		else if (aaa == 1 || bbb == 1){
			generalParams.edges_in_upperhem[i] = 0;
		}
		else{
			generalParams.edges_in_upperhem[i] = -1;
		}
		
	}*/

	int true_num_edges_in_upperhem = 0;
	for (int i = 0; i < coordInfoVecs.num_edges; i++){
		if (generalParams.edges_in_upperhem_list[i] != INT_MAX && generalParams.edges_in_upperhem_list[i] >= 0){
		true_num_edges_in_upperhem += 1;
		}
	}
	
	//std::vector<int> edge_to_ljparticle;
	//generalParams.edge_to_ljparticle.reserve(coordInfoVecs.num_edges);
	for (int i = 0; i < coordInfoVecs.num_edges; i++){
		generalParams.edge_to_ljparticle.push_back(-1);
	};
	/////////////////////////////////////////////////////////////////////
	////////////// END OF IDENTIFYING REG. WITH DIFF. MECH PROP /////////
	/////////////////////////////////////////////////////////////////////

	ComputeVolume(
		generalParams,
		coordInfoVecs,
		linearSpringInfoVecs,
		ljInfoVecs
	);
	double initial_volume;
	// initial_volume = generalParams.true_current_total_volume;
	// generalParams.eq_total_volume = generalParams.true_current_total_volume*VOLUME_FACTOR;//This is for setting different equilibrium volume to mimic growth or shirnkage.
	// std::cout<<"true current total volume = "<<generalParams.true_current_total_volume<<std::endl;
	// std::cout<<"eq total volume = "<<generalParams.eq_total_volume<<std::endl;

	//////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	////////////////////////////// START OF ACTUAL SIMULATION /////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////////////////////////

	/* Build the initial gradient weakend scale */
	dtb = 0.0;//dtb := distance to boundary
	generalParams.septin_ring_z = 0.0;
	generalParams.boundary_z = 0.0;
	//for (int k = 0; k < boundary_edge_list.size(); k++){
	for (int k = 0; k < boundary_node_list.size(); k++){
		double n1 = boundary_node_list[k];//coordInfoVecs.edges2Nodes_1[boundary_edge_list[k]];
		//double n2 = coordInfoVecs.edges2Nodes_2[boundary_edge_list[k]];
		//double cent_of_edge_x = (coordInfoVecs.nodeLocX[n1] + coordInfoVecs.nodeLocX[n2])/2.0;
		//double cent_of_edge_y = (coordInfoVecs.nodeLocY[n1] + coordInfoVecs.nodeLocY[n2])/2.0;
		//double cent_of_edge_z = (coordInfoVecs.nodeLocZ[n1] + coordInfoVecs.nodeLocZ[n2])/2.0;
		double dist_x = coordInfoVecs.nodeLocX[max_height_index] - coordInfoVecs.nodeLocX[n1];//cent_of_edge_x;
		double dist_y = coordInfoVecs.nodeLocY[max_height_index] - coordInfoVecs.nodeLocY[n1];//cent_of_edge_y;
		double dist_z = coordInfoVecs.nodeLocZ[max_height_index] - coordInfoVecs.nodeLocZ[n1];//cent_of_edge_z;
		// double temp_dist = sqrt((coordInfoVecs.nodeLocX[max_height_index] - coordInfoVecs.nodeLocX[n1])*(coordInfoVecs.nodeLocX[max_height_index] - coordInfoVecs.nodeLocX[n1]) +
		// (coordInfoVecs.nodeLocY[max_height_index] - coordInfoVecs.nodeLocY[n1])*(coordInfoVecs.nodeLocY[max_height_index] - coordInfoVecs.nodeLocY[n1]) +
		// 	(coordInfoVecs.nodeLocZ[max_height_index] - coordInfoVecs.nodeLocZ[n1])*(coordInfoVecs.nodeLocZ[max_height_index] - coordInfoVecs.nodeLocZ[n1]));
		double temp_dist = sqrt(dist_x*dist_x + dist_y*dist_y + dist_z*dist_z);
		generalParams.septin_ring_z += coordInfoVecs.nodeLocZ[n1];
		if (temp_dist >= dtb){
			dtb = temp_dist;
			/* "dtb" will be used to identify where the septin ring is located, and used to determine the Hill coefficient*/
		}
	}
	std::cout<<"dtb = "<<dtb<<std::endl;
	dtb_max = dtb + (generalParams.Rmin);
	
	std::cout<<"initial distance between cell tip and the boundary of weakened area = "<<dtb<<std::endl;
	std::cout<<"Notice that here, the distance from the tip to the boundary is slightly extended by half of the equilibrium length of an edge"<<std::endl;
	//std::cout<<"If this message is present, we are forcing a fixed portion of the bud tip to be occupied by the max concentration"<<std::endl;
	//generalParams.hilleqnconst = (dtb + generalParams.Rmin/4.0)/dtb_max;
	if (SCALE_TYPE == 4){
		generalParams.hilleqnconst = dtb/dtb_max;
		generalParams.hilleqnpow = 70.0;
		std::cout<<"hill equation constant K = "<<generalParams.hilleqnconst<<std::endl;
		std::cout<<"hill (equation) coefficient = "<<generalParams.hilleqnpow<<std::endl;
	}

	/////////////////////////////////////////////////////////////////
	////////// Building gradient weakening data struct //////////////
	////////// Building noes2Triangle data structure ////////////////
	/////////////////////////////////////////////////////////////////
	// Gradient-like scaling for each node & triangle. Even if we won't be using it, it is okay
	// to build it once. 'sigma' is currently unused in the actual function unless we are dealing
	// with SCALE_TYpe = 2 case.
	edgeswap_ptr->transferDtoH(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);
	edgeswap_ptr->gradient_weakening_update_host_vecs(sigma,
		//max_height_index,
		coordInfoVecs.nodeLocX[max_height_index],
		coordInfoVecs.nodeLocY[max_height_index],
		coordInfoVecs.nodeLocZ[max_height_index],
		dtb,
		dtb_max,
		generalParams,
		coordInfoVecs,
		build_ptr->hostSetInfoVecs);
	for (int u = 0; u < generalParams.maxNodeCount; u++){
		int BETA = edgeswap_ptr->nodes2Triangles_host_vecs(
			u,
			build_ptr->hostSetInfoVecs,
			coordInfoVecs,
			generalParams,
			auxVecs);
	}
	edgeswap_ptr->transferHtoD(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);
	//////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////
	//////////////////////////////////////////////////////////////////
	
	while (runSim == true){
		
		double current_time = 0.0;

		int translate_counter = 0;
		
		// We first allow the whole system to reach a quasi-steady state without edge-swap or weakening
        while (current_time < 3000.0*(Max_Runtime)){
            translate_counter += 1;
            Solve_Forces();
            double beta;
                
            AdvancePositions(
                coordInfoVecs,
                generalParams,
                domainParams);
                        
            new_total_energy = linearSpringInfoVecs.linear_spring_energy + 
                areaTriangleInfoVecs.area_triangle_energy + 
                bendingTriangleInfoVecs.bending_triangle_energy;// + 
                0.5*energy_rep;// + 
                //ljInfoVecs.lj_energy_M +
                //ljInfoVecs.lj_energy_LJ +
                //generalParams.volume_energy;

            // energy_gradient = sqrt((new_total_energy - old_total_energy)*(new_total_energy - old_total_energy))/old_total_energy;
            // if (current_time >= Max_Runtime*minimal_run_time_ratio && (energy_gradient/generalParams.dt) < energy_gradient_threshold){
            //     break;
            //     }
            old_total_energy = new_total_energy;
            current_time+=generalParams.dt;
        }

		std::cout<<"Time used for 'steady state' initial condition before growth and edge swaps = "<<current_time<<std::endl;
		std::cout<<"current total energy (before growth and edge swaps) = "<<new_total_energy<<std::endl;
		std::cout<<"LINEAR ENERGY = "<<linearSpringInfoVecs.linear_spring_energy<<std::endl;
		std::cout<<"BEND ENERGY = "<<bendingTriangleInfoVecs.bending_triangle_energy<<std::endl;
		std::cout<<"AREA ENERGY = "<<areaTriangleInfoVecs.area_triangle_energy<<std::endl;
		//std::cout<<"REPULSION ENERGY = "<<energy_rep<<std::endl;
		std::cout<<"VOLUME ENERGY = "<<generalParams.volume_energy<<std::endl;
		std::cout<<"true_current_total_volume (before growth and edge swaps) = "<<generalParams.true_current_total_volume<<std::endl;
		// std::cout<<"eq_total_volume = "<<generalParams.eq_total_volume<<std::endl;
		std::cout<<"current KBT = "<<generalParams.kT<<std::endl;
		if (isnan(new_total_energy)==1){
			std::cout<<"Nan or Inf position update !!!!"<<std::endl;
			runSim = false;
			break;
		}

		// Calculate current bud surface area
		double current_bud_area = 0.0;
		for (int k = 0; k < coordInfoVecs.num_triangles; k++){
			if (coordInfoVecs.triangles2Nodes_1[k] >= (INT_MAX - 1000.0) || coordInfoVecs.triangles2Nodes_1[k] <= (-INT_MAX + 1000.0) ||
				coordInfoVecs.triangles2Nodes_2[k] >= (INT_MAX - 1000.0) || coordInfoVecs.triangles2Nodes_2[k] <= (-INT_MAX + 1000.0) ||
				coordInfoVecs.triangles2Nodes_3[k] >= (INT_MAX - 1000.0) || coordInfoVecs.triangles2Nodes_3[k] <= (-INT_MAX + 1000.0)){
						continue;
					}
			else{
				if (generalParams.triangles_in_upperhem[k] == 1){
					double r1x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_1[k]];
					double r1y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_1[k]];
					double r1z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_1[k]];
					double r2x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_2[k]];
					double r2y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_2[k]];
					double r2z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_2[k]];
					double r3x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_3[k]];
					double r3y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_3[k]];
					double r3z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_3[k]];
					double norm_r1r2 = sqrt((r2x-r1x)*(r2x-r1x) + (r2y-r1y)*(r2y-r1y) + (r2z-r1z)*(r2z-r1z));
					double norm_r2r3 = sqrt((r3x-r2x)*(r3x-r2x) + (r3y-r2y)*(r3y-r2y) + (r3z-r2z)*(r3z-r2z));
					double norm_r3r1 = sqrt((r3x-r1x)*(r3x-r1x) + (r3y-r1y)*(r3y-r1y) + (r3z-r1z)*(r3z-r1z));
					double s = (norm_r1r2 + norm_r2r3 + norm_r3r1)/2.0;
					double area = sqrt(s*(s-norm_r1r2)*(s-norm_r2r3)*(s-norm_r3r1));
					current_bud_area += area;
				}
			}
		}
		double Initial_Bud_Area = current_bud_area;
		std::cout<<"Initial bud surface area (before growth and edge swaps) = "<<Initial_Bud_Area<<std::endl;

		//////////////////////////////////////////////////////////////////////////////////////////////////////////
		//This is where the change of mechanical properties and line tension activation officially starts.////////
		//////////////////////////////////////////////////////////////////////////////////////////////////////////
		generalParams.volume_spring_constant = 0.2;//(1.0/3.0)*areaTriangleInfoVecs.initial_area*1.0;
		std::cout<<"spring constant for surface normal expansion (pressure within the cell) = "<<generalParams.volume_spring_constant<<std::endl;
		generalParams.line_tension_constant = 50.0;//250.0;
		std::cout<<"spring constant for the septin ring = "<<generalParams.line_tension_constant<<std::endl;
		generalParams.length_scale = 1.0;//0.85;//0.1577;//1.0*generalParams.Rmin;// 0.8333;
		//std::cout<<"equilibrium length of each segment of the septin ring = "<<generalParams.length_scale<<std::endl;

		double scale_linear = linearSpringInfoVecs.spring_constant*0.75;//0.25;//25.0/2.5;//75.0/15.0;
		double scale_bend = bendingTriangleInfoVecs.spring_constant*0.1;//0.05;//10.0/1.0;//75.0/7.5;
		double scale_area = areaTriangleInfoVecs.spring_constant*0.75;//0.25;//50.0/5.0;//75.0/15.0;
		std::cout<<"weakened region linear = "<<scale_linear<<std::endl;
		std::cout<<"weakened region bend = "<<scale_bend<<std::endl;
		std::cout<<"weakened region area = "<<scale_area<<std::endl;
		//linearSpringInfoVecs.spring_constant_weak = linearSpringInfoVecs.spring_constant/scale_linear;
		//bendingTriangleInfoVecs.spring_constant_weak = bendingTriangleInfoVecs.spring_constant/scale_bend;
		//areaTriangleInfoVecs.spring_constant_weak = areaTriangleInfoVecs.spring_constant/scale_area;
		linearSpringInfoVecs.spring_constant_weak = scale_linear;
		bendingTriangleInfoVecs.spring_constant_weak = scale_bend;
		areaTriangleInfoVecs.spring_constant_weak = scale_area;
		//Scaling of the weakend mechanical properties.
		initial_volume = generalParams.true_current_total_volume;
		generalParams.eq_total_volume = generalParams.true_current_total_volume*VOLUME_FACTOR;//This is for setting different equilibrium volume.
		std::cout<<"true current total volume = "<<generalParams.true_current_total_volume<<std::endl;
		std::cout<<"eq total volume = "<<generalParams.eq_total_volume<<std::endl;
	
		storage->print_VTK_File();//Print the first animation output VTK, depicting the quasi-steady state of the system before
								  //edge-swap and so on.

		int edgeswap_iteration = 0;//Start counting how many times edge-swap algorithm is triggered.
		num_edge_loop = 0;//Reset the num_edge_loop to zero again here. Not entirely necessary but just to make sure.
		int LINE_TENSION_START = 0;
		
		bool WEAKENED_START = false;
		bool EDGESWAP_ALGORITHM_TRIGGERED;
		int number_of_simulation_step = 0;
 		while (initial_kT > 0){
			if (edgeswap_iteration >= NKBT){
				runSim = false;
				initial_kT = -1;
				break;
			}
			double VOLUME_RATIO = generalParams.true_current_total_volume/generalParams.eq_total_volume;
			
			//Rescale the equilibrium length for segments used for line tension so it is proportional to the cell size, if needed.
			if (generalParams.true_current_total_volume/initial_volume >= LINE_TENSION_THRESHOLD && edgeswap_iteration == 0){
			// 	if (LINE_TENSION_START < 1){
				double DIST = 0.0;
				double COUNT = 0.0;
				for (int t = 0; t < coordInfoVecs.num_edges; t++){
					if (generalParams.boundaries_in_upperhem[t] == 1){
						COUNT += 1.0;
						int node1 = coordInfoVecs.edges2Nodes_1[t];
						int node2 = coordInfoVecs.edges2Nodes_2[t];
						DIST += sqrt((coordInfoVecs.nodeLocX[node2] - coordInfoVecs.nodeLocX[node1])*(coordInfoVecs.nodeLocX[node2] - coordInfoVecs.nodeLocX[node1]) +
						(coordInfoVecs.nodeLocY[node2] - coordInfoVecs.nodeLocY[node1])*(coordInfoVecs.nodeLocY[node2] - coordInfoVecs.nodeLocY[node1]) + 
						(coordInfoVecs.nodeLocZ[node2] - coordInfoVecs.nodeLocZ[node1])*(coordInfoVecs.nodeLocZ[node2] - coordInfoVecs.nodeLocZ[node1]));
					}
				}
				// for (int t = 0; t < coordInfoVecs.num_edges; t++){
				// 	if (generalParams.boundaries_in_upperhem[t] == 1){
				// 		COUNT += 1.0;
				// 	}
				// }
				generalParams.length_scale = (DIST/COUNT)/generalParams.Rmin;
				std::cout<<"equilibrium length of each segment of the septin ring = "<<generalParams.length_scale*generalParams.Rmin<<std::endl;
				generalParams.eq_total_boundary_length = COUNT*generalParams.length_scale* generalParams.Rmin;
				std::cout<<"equilibrium length of the septin ring = "<<generalParams.eq_total_boundary_length<<std::endl;
				LINE_TENSION_START += 1;
			// 	}
				
			}
			
			current_time = 0.0;
			translate_counter = 0;
			EDGESWAP_ALGORITHM_TRIGGERED = false;
			bool end_of_relaxation = false;
			while (current_time < Max_Runtime){
				number_of_simulation_step += 1;
				if (Max_Runtime <= 0.0){
					std::cout<<"Max_Runtime is set to be 0 or negative! "<<std::endl;
					runSim = false;
					initial_kT = -1;
					break;
				}
					
				Solve_Forces();
					
				if (LINE_TENSION_START >= 1){
					ComputeLineTensionSprings(
						generalParams,
						coordInfoVecs,
						linearSpringInfoVecs);
					}
				
				/*energy_rep =
				ComputeMemRepulsionEnergy(
					coordInfoVecs,
					linearSpringInfoVecs, 
					capsidInfoVecs,
					generalParams,
						auxVecs);*/	

				///////////////////////////////////////////////////////////////////////////////////////////////
				// Updating nucleus (or endocytosis particles) based on its interaction ///////////////////////
				// with cell surface and other nucleus particles //////////////////////////////////////////////
				///////////////////////////////////////////////////////////////////////////////////////////////
				/*for (int i = 0; i < ljInfoVecs.LJ_PosX_all.size(); i++){

					ljInfoVecs.LJ_PosX = ljInfoVecs.LJ_PosX_all[i];
					// std::cout<<"LJ_PosX = "<<ljInfoVecs.LJ_PosX<<std::endl;
					ljInfoVecs.LJ_PosY = ljInfoVecs.LJ_PosY_all[i];
					// std::cout<<"LJ_PosY = "<<ljInfoVecs.LJ_PosY<<std::endl;
					ljInfoVecs.LJ_PosZ = ljInfoVecs.LJ_PosZ_all[i];
					
					ComputeLJSprings(
						coordInfoVecs,
						ljInfoVecs,
						generalParams);
					ljInfoVecs.forceX_all[i] =  ljInfoVecs.forceX;
					ljInfoVecs.forceY_all[i] =  ljInfoVecs.forceY;
					ljInfoVecs.forceZ_all[i] =  ljInfoVecs.forceZ;						

					ComputeLJSprings_LJ(
						coordInfoVecs,
						ljInfoVecs,
						generalParams);
					ljInfoVecs.forceX_all[i] +=  ljInfoVecs.forceX;
					ljInfoVecs.forceY_all[i] +=  ljInfoVecs.forceY;
					ljInfoVecs.forceZ_all[i] +=  ljInfoVecs.forceZ;
				}

				double beta;
				for (int i = 0; i < ljInfoVecs.LJ_PosX_all.size(); i++){
					
					if(nucleus_in_upperhem[i] == 1){
						beta = beta1;
					}
					else{
						beta = beta2;
					}
					ljInfoVecs.LJ_PosX_all[i] = ljInfoVecs.LJ_PosX_all[i] + generalParams.dt * ljInfoVecs.forceX_all[i];
					ljInfoVecs.LJ_PosY_all[i] = ljInfoVecs.LJ_PosY_all[i] + generalParams.dt * ljInfoVecs.forceY_all[i];
					ljInfoVecs.LJ_PosZ_all[i] = ljInfoVecs.LJ_PosZ_all[i] + generalParams.dt * (ljInfoVecs.forceZ_all[i] + beta);
				
				}*/	
				/////////////////////////////////////////////////////////////////////////////
				///////////////////////////////////////////////////////////////////////
				//////////////////////////////////////////////////////////////////				

				AdvancePositions(
					coordInfoVecs,
					generalParams,
					domainParams);

				new_total_energy = linearSpringInfoVecs.linear_spring_energy + 
				areaTriangleInfoVecs.area_triangle_energy + 
				bendingTriangleInfoVecs.bending_triangle_energy;// +
				// 0.5*energy_rep;

				energy_gradient = sqrt((new_total_energy - old_total_energy)*(new_total_energy - old_total_energy))/old_total_energy;
				old_total_energy = new_total_energy;
				current_time+=generalParams.dt;	
			
				if (translate_counter % translate_frequency == 0){
					newcenterX = 0.0;
					newcenterY = 0.0;
					newcenterZ = 0.0;
								
					for (int i = 0; i < generalParams.maxNodeCount; i++){//for (int i = 0; i < coordInfoVecs.nodeLocX.size(); i++){
						//std::cout<<i<<std::endl;
						newcenterX += coordInfoVecs.nodeLocX[i];
						//std::cout<<newcenterX<<std::endl;
						newcenterY += coordInfoVecs.nodeLocY[i];
						//std::cout<<newcenterY<<std::endl;
						newcenterZ += coordInfoVecs.nodeLocZ[i];
						//std::cout<<newcenterZ<<std::endl;
					}
				
					newcenterX = newcenterX/generalParams.maxNodeCount; //coordInfoVecs.nodeLocX.size();
					newcenterY = newcenterY/generalParams.maxNodeCount; //coordInfoVecs.nodeLocX.size();
					newcenterZ = newcenterZ/generalParams.maxNodeCount; //coordInfoVecs.nodeLocX.size();
					displacementX = newcenterX - generalParams.centerX;
					displacementY = newcenterY - generalParams.centerY;
					displacementZ = newcenterZ - generalParams.centerZ;
					
				
					for (int i = 0; i < generalParams.maxNodeCount; i++){
					coordInfoVecs.nodeLocX[i] += -displacementX;
					coordInfoVecs.nodeLocY[i] += -displacementY;
					coordInfoVecs.nodeLocZ[i] += -displacementZ;
					}
				
					for (int i = 0; i < ljInfoVecs.LJ_PosX_all.size(); i++){
						ljInfoVecs.LJ_PosX_all[i] += -displacementX;
						ljInfoVecs.LJ_PosY_all[i] += -displacementY;
						ljInfoVecs.LJ_PosZ_all[i] += -displacementZ;
					}

					ComputeVolume(
						generalParams,
						coordInfoVecs,
						linearSpringInfoVecs,
						ljInfoVecs);

				}
						

					
					// if (generalParams.SCALE_TYPE != 3){
					// 	if (translate_counter % (translate_frequency*1) == 0 || end_of_relaxation == true){
					// 		max_height = -10000.0;
					// 		double current_center_x = 0.0;
					// 		double current_center_y = 0.0;
							
					// 		for (int k = 0; k < generalParams.maxNodeCount; k++){
					// 			if (generalParams.nodes_in_upperhem[k] == 1){
					// 				current_center_x += coordInfoVecs.nodeLocX[k];
					// 				current_center_y += coordInfoVecs.nodeLocX[k];
					// 			}
								
					// 			if (coordInfoVecs. nodeLocZ[k] >= max_height){
					// 				max_height = coordInfoVecs.nodeLocZ[k];
					// 				max_height_index = k;
					// 			}
						
					// 		}
					// 		current_center_x = current_center_x/generalParams.maxNodeCount;
					// 		current_center_y = current_center_y/generalParams.maxNodeCount;
							
					// 		//std::cout<<"max_height_index = "<<max_height_index<<std::endl;
					// 		dtb = 0.0;//dtb := distance to boundary
					// 		generalParams.septin_ring_z = 0.0;
					// 		generalParams.boundary_z = 0.0;
					// 		//for (int k = 0; k < boundary_edge_list.size(); k++){
					// 		for (int k = 0; k < boundary_node_list.size(); k++){
					// 			double n1 = boundary_node_list[k];//coordInfoVecs.edges2Nodes_1[boundary_edge_list[k]];
					// 			//double n2 = coordInfoVecs.edges2Nodes_2[boundary_edge_list[k]];
					// 			//double cent_of_edge_x = (coordInfoVecs.nodeLocX[n1] + coordInfoVecs.nodeLocX[n2])/2.0;
					// 			//double cent_of_edge_y = (coordInfoVecs.nodeLocY[n1] + coordInfoVecs.nodeLocY[n2])/2.0;
					// 			//double cent_of_edge_z = (coordInfoVecs.nodeLocZ[n1] + coordInfoVecs.nodeLocZ[n2])/2.0;
					// 			double dist_x = current_center_x - coordInfoVecs.nodeLocX[n1];//coordInfoVecs.nodeLocX[max_height_index] - coordInfoVecs.nodeLocX[n1];//cent_of_edge_x;
					// 			double dist_y = current_center_y - coordInfoVecs.nodeLocY[n1];//coordInfoVecs.nodeLocY[max_height_index] - coordInfoVecs.nodeLocY[n1];//cent_of_edge_y;
					// 			double dist_z = max_height - coordInfoVecs.nodeLocZ[n1];//coordInfoVecs.nodeLocZ[max_height_index] - coordInfoVecs.nodeLocZ[n1];//cent_of_edge_z;
					// 			// double temp_dist = sqrt((coordInfoVecs.nodeLocX[max_height_index] - coordInfoVecs.nodeLocX[n1])*(coordInfoVecs.nodeLocX[max_height_index] - coordInfoVecs.nodeLocX[n1]) +
					// 			// (coordInfoVecs.nodeLocY[max_height_index] - coordInfoVecs.nodeLocY[n1])*(coordInfoVecs.nodeLocY[max_height_index] - coordInfoVecs.nodeLocY[n1]) +
					// 			// 	(coordInfoVecs.nodeLocZ[max_height_index] - coordInfoVecs.nodeLocZ[n1])*(coordInfoVecs.nodeLocZ[max_height_index] - coordInfoVecs.nodeLocZ[n1]));
					// 			// generalParams.septin_ring_z += coordInfoVecs.nodeLocZ[n1];
					// 			double temp_dist = sqrt(dist_x*dist_x + dist_y*dist_y + dist_z*dist_z);
					// 			if (temp_dist >= dtb){
					// 				dtb = temp_dist;
					// 				/* "dtb" will be used to identify where the septin ring is located, and used to determine the Hill coefficient*/
					// 			}
					// 		}
					// 		//std::cout<<"dtb = "<<dtb<<std::endl;
					// 		generalParams.septin_ring_z = generalParams.septin_ring_z/boundary_node_list.size();
					// 		generalParams.boundary_z = generalParams.septin_ring_z - generalParams.Rmin;
					// 		/* dtb will be only calculated once so we can effectively keep the Hill eqn curve consistent with only horizontal shift */
					// 		dtb_max = dtb + (generalParams.Rmin);
					// 		// generalParams.septin_ring_z = 0.0;
					// 		// generalParams.boundary_z = 0.0;
					// 		// //for (int k = 0; k < boundary_edge_list.size(); k++){
					// 		// for (int k = 0; k < boundary_node_list.size(); k++){
					// 		// 	double n1 = boundary_node_list[k];//coordInfoVecs.edges2Nodes_1[boundary_edge_list[k]];
					// 		// 	generalParams.septin_ring_z += coordInfoVecs.nodeLocZ[n1];
					// 		// }
					// 		//generalParams.septin_ring_z = generalParams.septin_ring_z/boundary_node_list.size();
					// 		//generalParams.boundary_z = generalParams.septin_ring_z - generalParams.Rmin;
					// 		/* dtb will be only calculated once so we can effectively keep the Hill eqn curve consistent with only horizontal shift */
					
					// 		//generalParams.hilleqnconst = (dtb + generalParams.Rmin/4.0)/dtb_max;
					// 		generalParams.hilleqnconst = dtb/dtb_max;

					// 		edgeswap_ptr->transferDtoH(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);//Currently this is treated as a backup of coordInfoVecs
					// 		edgeswap_ptr->gradient_weakening_update_host_vecs(sigma,
					// 			current_center_x,
					// 			current_center_y,
					// 			max_height,
					// 			dtb,
					// 			dtb_max,
					// 			generalParams,
					// 			coordInfoVecs,
					// 			build_ptr->hostSetInfoVecs);
					// 		edgeswap_ptr->transferHtoD(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);//Currently this is treated as a backup of coordInfoVecs
					// 		}
					// }	

				}

				end_of_relaxation = true;
					
				if (end_of_relaxation == true){
					std::random_device rand_dev;
					// std::mt19937 generator2(rand_dev());
					std::mt19937 generator_edgeswap(rand_dev());
					ComputeVolume(
						generalParams,
						coordInfoVecs,
						linearSpringInfoVecs,
						ljInfoVecs);

					if ((generalParams.true_current_total_volume/initial_volume) < 0.6 || generalParams.true_current_total_volume/initial_volume >= MAX_VOLUME_RATIO){
						generalParams.true_num_edges = 0;
						for (int i = 0; i < coordInfoVecs.num_edges; i++){
							if (coordInfoVecs.edges2Nodes_1[i] != INT_MAX && coordInfoVecs.edges2Nodes_2[i] != INT_MAX){
								generalParams.true_num_edges += 1;
							}
						}
						storage-> print_VTK_File();
						//storage-> storeVariables();
						
						if (generalParams.true_current_total_volume/initial_volume < 0.6){
							std::cout<<"Cell over compression 60%"<<std::endl;
						}
						else if (generalParams.true_current_total_volume/initial_volume >= MAX_VOLUME_RATIO){
							std::cout<<"Target volume ratio exceeded. Current volume ratio = "<<generalParams.true_current_total_volume/initial_volume<<std::endl;
						}
						std::cout<<"Current number of edgeswap iteration performed at volume-related termination = "<<edgeswap_iteration<<std::endl;
						std::cout<<"Current number of simulation step at volume-related termination = "<<number_of_simulation_step<<std::endl;

						Max_Runtime = 0.0;
						runSim = false;
						initial_kT = -1;
						break;
					}
					double current_bud_area = 0.0;
					for (int k = 0; k < coordInfoVecs.num_triangles; k++){
						if (coordInfoVecs.triangles2Nodes_1[k] >= (INT_MAX - 1000.0) || coordInfoVecs.triangles2Nodes_1[k] <= (-INT_MAX + 1000.0) ||
							coordInfoVecs.triangles2Nodes_2[k] >= (INT_MAX - 1000.0) || coordInfoVecs.triangles2Nodes_2[k] <= (-INT_MAX + 1000.0) ||
							coordInfoVecs.triangles2Nodes_3[k] >= (INT_MAX - 1000.0) || coordInfoVecs.triangles2Nodes_3[k] <= (-INT_MAX + 1000.0)){
									continue;
								}
						else{
							if (generalParams.triangles_in_upperhem[k] == 1){
								double r1x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_1[k]];
								double r1y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_1[k]];
								double r1z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_1[k]];
								double r2x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_2[k]];
								double r2y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_2[k]];
								double r2z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_2[k]];
								double r3x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_3[k]];
								double r3y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_3[k]];
								double r3z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_3[k]];
								double norm_r1r2 = sqrt((r2x-r1x)*(r2x-r1x) + (r2y-r1y)*(r2y-r1y) + (r2z-r1z)*(r2z-r1z));
								double norm_r2r3 = sqrt((r3x-r2x)*(r3x-r2x) + (r3y-r2y)*(r3y-r2y) + (r3z-r2z)*(r3z-r2z));
								double norm_r3r1 = sqrt((r3x-r1x)*(r3x-r1x) + (r3y-r1y)*(r3y-r1y) + (r3z-r1z)*(r3z-r1z));
								double s = (norm_r1r2 + norm_r2r3 + norm_r3r1)/2.0;
								double area = sqrt(s*(s-norm_r1r2)*(s-norm_r2r3)*(s-norm_r3r1));
								current_bud_area += area;
							}
						}
					}
							// std::cout<<"Current bud surface area = "<<current_bud_area<<std::endl;
					if (current_bud_area/Initial_Bud_Area >= MAX_BUD_AREA_RATIO){
						std::cout<<"Target bud surface area ratio exceeded. Current bud surface area ratio = "<<current_bud_area/Initial_Bud_Area<<std::endl;
						std::cout<<"Current number of edgeswap iteration performed at area-related termination = "<<edgeswap_iteration<<std::endl;
						std::cout<<"Current number of simulation step at area-related termination = "<<number_of_simulation_step<<std::endl;
						generalParams.true_num_edges = 0;
						for (int i = 0; i < coordInfoVecs.num_edges; i++){
							if (coordInfoVecs.edges2Nodes_1[i] != INT_MAX && coordInfoVecs.edges2Nodes_2[i] != INT_MAX){
								generalParams.true_num_edges += 1;
							}
						}
						storage-> print_VTK_File();
						Max_Runtime = 0.0;
						runSim = false;
						initial_kT = -1;
						break;
					}

					edgeswap_ptr->transferDtoH(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);
						
					VectorShuffleForEdgeswapLoop.clear();
					for (int i = 0; i < coordInfoVecs.num_edges; i++){
						if (generalParams.edges_in_upperhem_list[i] >= 0 && 
							generalParams.edges_in_upperhem_list[i] != INT_MAX &&
							//generalParams.edges_in_upperhem[i] < coordInfoVecs.num_edges &&
							//generalParams.edges_in_upperhem[i] != -INT_MAX &&
							generalParams.boundaries_in_upperhem[i] != 1)
							// VectorShuffleForEdgeswapLoop.push_back(generalParams.edges_in_upperhem_list[i]);
							VectorShuffleForEdgeswapLoop.push_back(i);
						}	
				
					num_edge_loop = round(true_num_edges_in_upperhem*SAMPLE_SIZE);
					if (num_edge_loop <= min_num_edge_loop){
						num_edge_loop = min_num_edge_loop;
					}
						
					std::shuffle(std::begin(VectorShuffleForEdgeswapLoop), std::end(VectorShuffleForEdgeswapLoop), generator_edgeswap);
					for (int edge_loop = 0; edge_loop < num_edge_loop; edge_loop++) {
														
						std::uniform_int_distribution<int> distribution(1,VectorShuffleForEdgeswapLoop.size());
						
						int dice_roll = distribution(generator_edgeswap);
						
						int edge = VectorShuffleForEdgeswapLoop[dice_roll - 1];
						//int edge = dice_roll -1;
						while (generalParams.boundaries_in_upperhem[edge] == 1 || edge == INT_MAX || edge < 0){
							dice_roll = distribution(generator_edgeswap);
							
							int edge = VectorShuffleForEdgeswapLoop[dice_roll - 1];
							//edge =  generalParams.edges_in_upperhem_list[dice_roll - 1];
							//edge = dice_roll -1;
							}
						//int edge = generalParams.edges_in_upperhem_list[edge_loop];
						//int edge = VectorShuffleForEdgeswapLoop[edge_loop];
						// std::cout<<"edge = "<<edge<<std::endl;
						if (edge < 0 || edge == INT_MAX){
							continue;
						}

						int ALPHA = edgeswap_ptr->edge_swap_host_vecs(
							edge,
							generalParams,
							build_ptr->hostSetInfoVecs,
							linearSpringInfoVecs,
							bendingTriangleInfoVecs,
							areaTriangleInfoVecs);
						
					}
						
					edgeswap_ptr->transferHtoD(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);//Currently this is treated as a backup of coordInfoVecs
					
					EDGESWAP_ALGORITHM_TRIGGERED = true;
					edgeswap_iteration += 1;
					translate_counter += 1;
				}
				
				if (EDGESWAP_ALGORITHM_TRIGGERED == false){
					//std::cout<<"current_time = "<<current_time<<std::endl;
					std::cout<<"EDGE_SWAP IS TRIGGERED BECAUSE PREVIOUS RELAXATION STEPS SOMEHOW FAIL TO TRIGGER EDGESWAP NORMALLY. PLEASE INVESTIGATE."<<std::endl;
					runSim = false;
					initial_kT = -1;
					Max_Runtime = 0.0;
					break;
				}
					
				if (edgeswap_iteration % (GROWTH_FREQUENCY*GROWTH_FREQUENCY_SCALE) == 0){
						for (int v = 0; v < coordInfoVecs.num_edges; v++){
						double ev1 = coordInfoVecs.edges2Nodes_1[v];
						double ev2 = coordInfoVecs.edges2Nodes_2[v];
						if (ev1 == INT_MAX || ev2 == INT_MAX){
							continue;
						}
						double ed = sqrt((coordInfoVecs.nodeLocX[ev2] - coordInfoVecs.nodeLocX[ev1])*(coordInfoVecs.nodeLocX[ev2] - coordInfoVecs.nodeLocX[ev1]) +
									(coordInfoVecs.nodeLocY[ev2] - coordInfoVecs.nodeLocY[ev1])*(coordInfoVecs.nodeLocY[ev2] - coordInfoVecs.nodeLocY[ev1]) +
									(coordInfoVecs.nodeLocZ[ev2] - coordInfoVecs.nodeLocZ[ev1])*(coordInfoVecs.nodeLocZ[ev2] - coordInfoVecs.nodeLocZ[ev1]));
						if (ed >= 2.0){
							std::cout<<"Edge over extension, possibly some instability occuring. Aborting the simulation."<<std::endl;
							runSim = false;
							initial_kT = -1;
							break;
						}
					}
					// generalParams.angle_per_edge.clear();
					generalParams.true_num_edges = 0;
					for (int i = 0; i < coordInfoVecs.num_edges; i++){
						if (coordInfoVecs.edges2Nodes_1[i] != INT_MAX && coordInfoVecs.edges2Nodes_2[i] != INT_MAX){
							generalParams.true_num_edges += 1;
						}
						}
						storage->print_VTK_File();
						double current_bud_area = 0.0;
						for (int k = 0; k < coordInfoVecs.num_triangles; k++){
						if (coordInfoVecs.triangles2Nodes_1[k] >= (INT_MAX - 1000.0) || coordInfoVecs.triangles2Nodes_1[k] <= (-INT_MAX + 1000.0) ||
							coordInfoVecs.triangles2Nodes_2[k] >= (INT_MAX - 1000.0) || coordInfoVecs.triangles2Nodes_2[k] <= (-INT_MAX + 1000.0) ||
							coordInfoVecs.triangles2Nodes_3[k] >= (INT_MAX - 1000.0) || coordInfoVecs.triangles2Nodes_3[k] <= (-INT_MAX + 1000.0)){
									continue;
								}
						else{
							if (generalParams.triangles_in_upperhem[k] == 1){
								double r1x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_1[k]];
								double r1y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_1[k]];
								double r1z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_1[k]];
								double r2x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_2[k]];
								double r2y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_2[k]];
								double r2z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_2[k]];
								double r3x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_3[k]];
								double r3y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_3[k]];
								double r3z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_3[k]];
								double norm_r1r2 = sqrt((r2x-r1x)*(r2x-r1x) + (r2y-r1y)*(r2y-r1y) + (r2z-r1z)*(r2z-r1z));
								double norm_r2r3 = sqrt((r3x-r2x)*(r3x-r2x) + (r3y-r2y)*(r3y-r2y) + (r3z-r2z)*(r3z-r2z));
								double norm_r3r1 = sqrt((r3x-r1x)*(r3x-r1x) + (r3y-r1y)*(r3y-r1y) + (r3z-r1z)*(r3z-r1z));
								double s = (norm_r1r2 + norm_r2r3 + norm_r3r1)/2.0;
								double area = sqrt(s*(s-norm_r1r2)*(s-norm_r2r3)*(s-norm_r3r1));
								current_bud_area += area;
							}
						}
						}
						std::cout<<"Current bud surface area = "<<current_bud_area<<std::endl;
						std::cout<<"Current number of edgeswap performed = "<<edgeswap_iteration<<std::endl;
					//  std::cout<<"current Hill equation constant = "<<generalParams.hilleqnconst<<std::endl;
						//storage->storeVariables();
						std::cout<<"current total energy = "<< new_total_energy<<std::endl;
					//  std::cout<<"LINEAR ENERGY = "<<linearSpringInfoVecs.linear_spring_energy<<std::endl;
					// std::cout<<"BEND ENERGY = "<<bendingTriangleInfoVecs.bending_triangle_energy<<std::endl;
					// std::cout<<"AREA ENERGY = "<<areaTriangleInfoVecs.area_triangle_energy<<std::endl;
					//std::cout<<"REPULSION ENERGY = "<<energy_rep<<std::endl;
					// std::cout<<"VOLUME ENERGY = "<<generalParams.volume_energy<<std::endl;
						std::cout<<"energy_gradient = "<<energy_gradient<<std::endl;
						std::cout<<"true current total volume = "<<generalParams.true_current_total_volume<<std::endl;
					std::cout<<"equilibrium total volume = "<<generalParams.eq_total_volume<<std::endl;
				}
				if (edgeswap_iteration == NKBT-1 ){
					//storage->storeVariables();
					}


/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////// GROWTH OF THE CELL (MEMBRANE) ////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////	
//GROWTH_COUNTER = 0;
if (edgeswap_iteration % GROWTH_FREQUENCY == 0){
	GROWTH_COUNTER += 1;

//This commented out section is for the case with temporal (or volume-based) restoration of mechanical properties.
/*		double MAX_VOLUME_RATIO_stiffening = 1.0;
		double delayed_stiffening = 1.0;
	if (edgeswap_iteration <= GROWTH_FREQUENCY){
		std::cout<<"delayed_stiffening = "<<delayed_stiffening<<std::endl;
		std::cout<<"This determines when stiffening actually start taking effect after the cell volume gain (X*100)% increase"<<std::endl;
	}
// linearSpringInfoVecs.spring_constant_weak = linearSpringInfoVecs.spring_constant_weak +
// 								((linearSpringInfoVecs.spring_constant - scale_linear)/500.0);//(MAX_GROWTH_EVENT_NUMBER));
if (generalParams.true_current_total_volume >= initial_volume*delayed_stiffening){
	linearSpringInfoVecs.spring_constant_weak = scale_linear +
									((linearSpringInfoVecs.spring_constant - scale_linear)*(generalParams.true_current_total_volume - initial_volume*delayed_stiffening)/(initial_volume*MAX_VOLUME_RATIO_stiffening - initial_volume*delayed_stiffening));

	if (linearSpringInfoVecs.spring_constant_weak >= linearSpringInfoVecs.spring_constant){
		linearSpringInfoVecs.spring_constant_weak = linearSpringInfoVecs.spring_constant;
	}
}

// bendingTriangleInfoVecs.spring_constant_weak = bendingTriangleInfoVecs.spring_constant_weak +
// 								((bendingTriangleInfoVecs.spring_constant - scale_bend)/500.0);///MAX_GROWTH_EVENT_NUMBER);;

if (generalParams.true_current_total_volume >= initial_volume*delayed_stiffening){
	bendingTriangleInfoVecs.spring_constant_weak = scale_bend +
									((bendingTriangleInfoVecs.spring_constant - scale_bend)*(generalParams.true_current_total_volume - initial_volume*delayed_stiffening)/(initial_volume*MAX_VOLUME_RATIO_stiffening - initial_volume*delayed_stiffening));///MAX_GROWTH_EVENT_NUMBER);;

	if (bendingTriangleInfoVecs.spring_constant_weak >= bendingTriangleInfoVecs.spring_constant){
		bendingTriangleInfoVecs.spring_constant_weak = bendingTriangleInfoVecs.spring_constant;
	}
}
// areaTriangleInfoVecs.spring_constant_weak = areaTriangleInfoVecs.spring_constant_weak +
// 								((areaTriangleInfoVecs.spring_constant - scale_area)/500.0);///MAX_GROWTH_EVENT_NUMBER);;

if (generalParams.true_current_total_volume >= initial_volume*delayed_stiffening){
	areaTriangleInfoVecs.spring_constant_weak = scale_area +
									((areaTriangleInfoVecs.spring_constant - scale_area)*(generalParams.true_current_total_volume - initial_volume*delayed_stiffening)/(initial_volume*MAX_VOLUME_RATIO_stiffening - initial_volume*delayed_stiffening));///MAX_GROWTH_EVENT_NUMBER);;

	if (areaTriangleInfoVecs.spring_constant_weak >= areaTriangleInfoVecs.spring_constant){
		areaTriangleInfoVecs.spring_constant_weak = areaTriangleInfoVecs.spring_constant;
	}
}
// generalParams.kT = generalParams.kT + (0.07 - 0.035)/500.0;//MAX_GROWTH_EVENT_NUMBER;
if (generalParams.true_current_total_volume >= initial_volume*delayed_stiffening){
	generalParams.kT = initial_kT + (initial_kT*2.0 - initial_kT)*((generalParams.true_current_total_volume - initial_volume*delayed_stiffening)/(initial_volume*MAX_VOLUME_RATIO_stiffening - initial_volume*delayed_stiffening));
	if (generalParams.kT >= initial_kT*2.0){
		generalParams.kT = initial_kT*2.0;
	}
}
if (edgeswap_iteration % (2*GROWTH_FREQUENCY) == 0){
	std::cout<<"kT = "<<generalParams.kT<<std::endl;
	std::cout<<"linear spring coeff weak = "<<linearSpringInfoVecs.spring_constant_weak<<std::endl;
	std::cout<<"bending spring coeff weak = "<<bendingTriangleInfoVecs.spring_constant_weak<<std::endl;
	std::cout<<"area spring coeff weak = "<<areaTriangleInfoVecs.spring_constant_weak<<std::endl;
	std::cout<<"true current total volume = "<<generalParams.true_current_total_volume<<std::endl;
	std::cout<<"initial volume = "<<initial_volume<<std::endl;
	std::cout<<"initial volume * MAX_VOLUME RATIO = "<<initial_volume*MAX_VOLUME_RATIO<<std::endl;
	std::cout<<"current volume ratio = "<<(generalParams.true_current_total_volume)/(initial_volume*MAX_VOLUME_RATIO)<<std::endl;
}
// if (generalParams.true_current_total_volume >= initial_volume*delayed_stiffening){
// 	generalParams.strain_threshold = 0.05 +
// 								((0.4 - 0.05)*pow((generalParams.true_current_total_volume - initial_volume*delayed_stiffening)/(initial_volume*MAX_VOLUME_RATIO_stiffening - initial_volume*delayed_stiffening), 1.0));///MAX_GROWTH_EVENT_NUMBER)
// }

max_height = -10000.0;
double current_center_x = 0.0;
double current_center_y = 0.0;

for (int k = 0; k < generalParams.maxNodeCount; k++){
	if (generalParams.nodes_in_upperhem[k] == 1){
		current_center_x += coordInfoVecs.nodeLocX[k];
		current_center_y += coordInfoVecs.nodeLocX[k];
	}
	
	if (coordInfoVecs. nodeLocZ[k] >= max_height){
		max_height = coordInfoVecs.nodeLocZ[k];
		max_height_index = k;
	}

}
current_center_x = current_center_x/generalParams.maxNodeCount;
current_center_y = current_center_y/generalParams.maxNodeCount;
// double bdry_to_tip = 0.0;
double bdry_to_tip_height = 0.0;
for (int y = 0; y < boundary_edge_list.size(); y++){
	// double edge_mdpt_x = (coordInfoVecs.nodeLocX[coordInfoVecs.edges2Nodes_1[boundary_edge_list[y]]] +
	// 						coordInfoVecs.nodeLocX[coordInfoVecs.edges2Nodes_2[boundary_edge_list[y]]])/2.0;
	// double edge_mdpt_y = (coordInfoVecs.nodeLocY[coordInfoVecs.edges2Nodes_1[boundary_edge_list[y]]] +
	// 						coordInfoVecs.nodeLocY[coordInfoVecs.edges2Nodes_2[boundary_edge_list[y]]])/2.0;
	double edge_mdpt_z = (coordInfoVecs.nodeLocZ[coordInfoVecs.edges2Nodes_1[boundary_edge_list[y]]] +
							coordInfoVecs.nodeLocZ[coordInfoVecs.edges2Nodes_2[boundary_edge_list[y]]])/2.0;
	// bdry_to_tip += sqrt(pow(current_center_x - edge_mdpt_x,2.0)+pow(current_center_y - edge_mdpt_y,2.0)+pow(coordInfoVecs.nodeLocZ[max_height_index] - edge_mdpt_z,2.0));
	bdry_to_tip_height += sqrt(pow(coordInfoVecs.nodeLocZ[max_height_index] - edge_mdpt_z,2.0));
}
// bdry_to_tip = bdry_to_tip/boundary_edge_list.size();
bdry_to_tip_height = bdry_to_tip_height/boundary_edge_list.size();*/

VectorShuffleForGrowthLoop.clear();
int VectorShuffleForGrowthLoop_COUNT = 0;
for (int y = 0; y < coordInfoVecs.num_edges; y++){
	if (generalParams.edges_in_upperhem_list[y] >= 0 &&
		generalParams.edges_in_upperhem_list[y] != INT_MAX &&
		generalParams.boundaries_in_upperhem[y] != 1){
		VectorShuffleForGrowthLoop.push_back(y);
		VectorShuffleForGrowthLoop_COUNT += 1;
	}
	/*if (generalParams.edges_in_upperhem_list[y] >= 0 &&
		generalParams.edges_in_upperhem_list[y] != INT_MAX &&
		generalParams.boundaries_in_upperhem[y] != 1 &&
		edges_in_growth[y] == 1){
		VectorShuffleForGrowthLoop.push_back(y);
	}*/
	
	
}

// for (int y = 0; y < coordInfoVecs.num_edges; y++){
// 	// std::cout<<y<<std::endl;
// 	if (generalParams.edges_in_upperhem_list[y] >= 0 &&
// 		generalParams.edges_in_upperhem_list[y] != INT_MAX &&
// 		generalParams.edges_in_upperhem_list[y] <= (INT_MAX-1000) &&
// 		generalParams.edges_in_upperhem_list[y] >= (-INT_MAX+1000) &&
// 		generalParams.boundaries_in_upperhem[y] != 1){
// 			// std::cout<<"IF condition satisfied"<<std::endl;
// 			// std::cout<<"generalParams.edges_in_upperhem_list = "<<generalParams.edges_in_upperhem_list[y]<<std::endl;
// 			if (coordInfoVecs.edges2Nodes_1[y] < 0 || coordInfoVecs.edges2Nodes_1[y] >= (INT_MAX-1000)){
// 				continue;
// 			}
// 			else if (coordInfoVecs.edges2Nodes_2[y] < 0 || coordInfoVecs.edges2Nodes_2[y] >= (INT_MAX-1000)){
// 				continue;
// 			}
// 			// double edge_mdpt_x = (coordInfoVecs.nodeLocX[coordInfoVecs.edges2Nodes_1[y]] +
// 			// 					coordInfoVecs.nodeLocX[coordInfoVecs.edges2Nodes_2[y]])/2.0;
// 			// // std::cout<<edge_mdpt_x<<std::endl;
// 			// double edge_mdpt_y = (coordInfoVecs.nodeLocY[coordInfoVecs.edges2Nodes_1[y]] +
// 			// 						coordInfoVecs.nodeLocY[coordInfoVecs.edges2Nodes_2[y]])/2.0;
// 			// // std::cout<<edge_mdpt_y<<std::endl;
// 			double edge_mdpt_z = (coordInfoVecs.nodeLocZ[coordInfoVecs.edges2Nodes_1[y]] +
// 									coordInfoVecs.nodeLocZ[coordInfoVecs.edges2Nodes_2[y]])/2.0;
// 			// std::cout<<edge_mdpt_z<<std::endl;
// 			// double current_edge_to_tip = sqrt(pow(current_center_x - edge_mdpt_x,2.0)+pow(current_center_y - edge_mdpt_y,2.0)+pow(coordInfoVecs.nodeLocZ[max_height_index] - edge_mdpt_z,2.0));
// 			double current_edge_to_tip_height = sqrt(pow(coordInfoVecs.nodeLocZ[max_height_index] - edge_mdpt_z,2.0));
// 			// std::cout<<"current_edge_to_tip = "<<current_edge_to_tip<<std::endl;
// 		// if ((current_edge_to_tip/bdry_to_tip) <= 0.8 && bdry_to_tip >= (dtb*1.5)){
// 		if ((current_edge_to_tip_height/bdry_to_tip_height) <= portion_of_bud_for_wall_insertion && 
// 				bdry_to_tip_height >= (dtb*ratio_for_restricted_wall_insertion)){
// 			VectorShuffleForGrowthLoop.push_back(y);
// 			VectorShuffleForGrowthLoop_COUNT += 1;
// 		}
// 		// else if(bdry_to_tip < (dtb*1.5)){
// 		else if(bdry_to_tip_height < (dtb*ratio_for_restricted_wall_insertion)){
// 			VectorShuffleForGrowthLoop.push_back(y);
// 			VectorShuffleForGrowthLoop_COUNT += 1;
// 		}
// 	}
// 	/*if (generalParams.edges_in_upperhem_list[y] >= 0 &&
// 		generalParams.edges_in_upperhem_list[y] != INT_MAX &&
// 		generalParams.boundaries_in_upperhem[y] != 1 &&
// 		edges_in_growth[y] == 1){
// 		VectorShuffleForGrowthLoop.push_back(y);
// 	}*/
	
	
// }
std::cout<<VectorShuffleForGrowthLoop_COUNT<<std::endl;

std::random_device rand_dev;
std::mt19937 generator3(rand_dev());
std::shuffle(std::begin(VectorShuffleForGrowthLoop), std::end(VectorShuffleForGrowthLoop), generator3);
int MAX_GROWTH_TEST = VectorShuffleForGrowthLoop.size();
bool triggered = false;
int true_DELTA = 0;
int MAX_GROWTH_PER_GROWTH_EVENT = 1; //Max number of accepted growth per growth event
//std::cout<<"BEGIN GROWTH ALGORITHM"<<std::endl;
edgeswap_ptr->transferDtoH(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);
int GROWTH_COUNT = 0;
for (int p = 0; p < MAX_GROWTH_TEST; p++){
	if (coordInfoVecs.edges2Nodes_1[VectorShuffleForGrowthLoop[p]] < 0 || coordInfoVecs.edges2Nodes_1[VectorShuffleForGrowthLoop[p]] == INT_MAX){
		continue;
	}
	else if (coordInfoVecs.edges2Nodes_2[VectorShuffleForGrowthLoop[p]] < 0 || coordInfoVecs.edges2Nodes_2[VectorShuffleForGrowthLoop[p]] == INT_MAX){
		continue;
	}
	//std::cout<<"begin growth test"<<std::endl;
	int DELTA = edgeswap_ptr->growth_host_vecs(
		VectorShuffleForGrowthLoop[p],
		generalParams,
		build_ptr->hostSetInfoVecs,
		coordInfoVecs,
		linearSpringInfoVecs,
		bendingTriangleInfoVecs,
		areaTriangleInfoVecs);
	GROWTH_COUNT += DELTA;
	TOTAL_GROWTH_COUNTER += DELTA;
	if (GROWTH_COUNT >= MAX_GROWTH_PER_GROWTH_EVENT){
	    break;
	}
}
TOTAL_GROWTH_ATTEMPT += 1;
edgeswap_ptr->transferHtoD(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);
std::cout<<"number of cell wall insertion = "<<GROWTH_COUNT<<std::endl;
std::cout<<"Total growth event triggered = "<<TOTAL_GROWTH_COUNTER<<std::endl;
std::cout<<"Total growth event attempt = "<<TOTAL_GROWTH_ATTEMPT<<std::endl;
// if (TOTAL_GROWTH_COUNTER > NUMBER_OF_TARGETED_GROWTH_EVENT){
	// std::cout<<"Target number of growth event is reached. Terminate the simulation. Current cell volume to initial volume ratio = "<<generalParams.true_current_total_volume/initial_volume<<std::endl;
	// Max_Runtime = 0.0;
	// runSim = false;
	// initial_kT = -1;
	// break;
	// }


				if (triggered == true){	
					true_num_edges_in_upperhem = 0;
					for (int i = 0; i < coordInfoVecs.num_edges; i++){
						if (generalParams.edges_in_upperhem_list[i] != INT_MAX && generalParams.edges_in_upperhem_list[i] >= 0){
							true_num_edges_in_upperhem += 1;
							//break;
						}
					}
					//std::cout<<"WHERE iS THE PROBLEM 3"<<std::endl;
				}
			}
			
			
			
			
//std::cout<<"GROWTH DONE!"<<std::endl;
 ////storage->print_VTK_File();
////storage->storeVariables();

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////// END OF GROWTH SECTION //////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
					

ComputeVolume(
	generalParams,
	coordInfoVecs,
	linearSpringInfoVecs,
	ljInfoVecs);
					
					
 			}
		
		}
		

	};
	
	





void System::assignStorage(std::shared_ptr<Storage> _storage) {
	storage = _storage;
};
void System::set_weak_builder(std::weak_ptr<SystemBuilder> _weak_bld_ptr) {
	weak_bld_ptr = _weak_bld_ptr;
};



//initialize memory for thrust vectors and set coordInfoVecs vals from input. 
void System::initializeSystem(HostSetInfoVecs& hostSetInfoVecs) {
	std::cout<<"Initializing"<<std::endl;

	generalParams.maxNodeCount = hostSetInfoVecs.nodeLocX.size();
	coordInfoVecs.num_edges = hostSetInfoVecs.edges2Nodes_1.size();
	coordInfoVecs.num_triangles = hostSetInfoVecs.triangles2Nodes_1.size();

	std::cout<<"num nodes: "<< generalParams.maxNodeCount << std::endl;
	std::cout<<"num edges: "<< coordInfoVecs.num_edges << std::endl;
	std::cout<<"num elems: "<< coordInfoVecs.num_triangles << std::endl;
	//allocate memory
	int mem_prealloc = 3;
	coordInfoVecs.scaling_per_edge.resize(mem_prealloc*coordInfoVecs.num_edges, 0.0);
	hostSetInfoVecs.scaling_per_edge.resize(coordInfoVecs.scaling_per_edge.size(), 0.0);

	coordInfoVecs.isNodeFixed.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size(),false);
	coordInfoVecs.prevNodeLocX.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	coordInfoVecs.prevNodeLocY.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	coordInfoVecs.prevNodeLocZ.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());

	coordInfoVecs.prevNodeForceX.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	coordInfoVecs.prevNodeForceY.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	coordInfoVecs.prevNodeForceZ.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	
	coordInfoVecs.nodeLocX.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	coordInfoVecs.nodeLocY.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	coordInfoVecs.nodeLocZ.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());

	coordInfoVecs.nodeForceX.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size(), 0.0);
	coordInfoVecs.nodeForceY.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size(), 0.0);
	coordInfoVecs.nodeForceZ.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size(), 0.0);

	coordInfoVecs.triangles2Nodes_1.resize( mem_prealloc*coordInfoVecs.num_triangles );
	coordInfoVecs.triangles2Nodes_2.resize( mem_prealloc*coordInfoVecs.num_triangles );
	coordInfoVecs.triangles2Nodes_3.resize( mem_prealloc*coordInfoVecs.num_triangles );
	
	coordInfoVecs.triangles2Edges_1.resize( mem_prealloc*coordInfoVecs.num_triangles );
	coordInfoVecs.triangles2Edges_2.resize( mem_prealloc*coordInfoVecs.num_triangles );
	coordInfoVecs.triangles2Edges_3.resize( mem_prealloc*coordInfoVecs.num_triangles );

	coordInfoVecs.edges2Nodes_1.resize( mem_prealloc*coordInfoVecs.num_edges );
	coordInfoVecs.edges2Nodes_2.resize( mem_prealloc*coordInfoVecs.num_edges );
	
	coordInfoVecs.edges2Triangles_1.resize( mem_prealloc*coordInfoVecs.num_edges );
	coordInfoVecs.edges2Triangles_2.resize( mem_prealloc*coordInfoVecs.num_edges );

	coordInfoVecs.nndata1.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata2.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata3.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata4.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata5.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata6.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata7.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata8.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.nndata9.resize( mem_prealloc*generalParams.maxNodeCount);
	//coordInfoVecs.nndata10.resize( mem_prealloc*generalParams.maxNodeCount);
	//coordInfoVecs.nndata11.resize( mem_prealloc*generalParams.maxNodeCount);
	//coordInfoVecs.nndata12.resize( mem_prealloc*generalParams.maxNodeCount);

	coordInfoVecs.SurfaceNormalX.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.SurfaceNormalY.resize( mem_prealloc*generalParams.maxNodeCount);
	coordInfoVecs.SurfaceNormalZ.resize( mem_prealloc*generalParams.maxNodeCount);

	generalParams.nodes_in_upperhem.resize(mem_prealloc*generalParams.maxNodeCount);
	generalParams.triangles_in_upperhem.resize(mem_prealloc*coordInfoVecs.num_triangles);
	generalParams.edges_in_upperhem.resize(mem_prealloc*coordInfoVecs.num_edges);
	generalParams.edges_in_upperhem_list.resize(mem_prealloc*coordInfoVecs.num_edges);
	generalParams.boundaries_in_upperhem.resize(mem_prealloc*coordInfoVecs.num_edges, -1);

	hostSetInfoVecs.nodes_in_upperhem.resize(generalParams.nodes_in_upperhem.size());
	hostSetInfoVecs.triangles_in_upperhem.resize(generalParams.triangles_in_upperhem.size());
	hostSetInfoVecs.edges_in_upperhem.resize(generalParams.edges_in_upperhem.size());
	hostSetInfoVecs.edges_in_upperhem_list.resize(mem_prealloc*coordInfoVecs.num_edges);
	hostSetInfoVecs.boundaries_in_upperhem.resize(mem_prealloc*coordInfoVecs.num_edges, -1);

	hostSetInfoVecs.nodes2Triangles_1.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	hostSetInfoVecs.nodes2Triangles_2.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	hostSetInfoVecs.nodes2Triangles_3.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	hostSetInfoVecs.nodes2Triangles_4.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	hostSetInfoVecs.nodes2Triangles_5.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	hostSetInfoVecs.nodes2Triangles_6.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	hostSetInfoVecs.nodes2Triangles_7.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	hostSetInfoVecs.nodes2Triangles_8.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	hostSetInfoVecs.nodes2Triangles_9.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	

	coordInfoVecs.nodes2Triangles_1.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	coordInfoVecs.nodes2Triangles_2.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	coordInfoVecs.nodes2Triangles_3.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	coordInfoVecs.nodes2Triangles_4.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	coordInfoVecs.nodes2Triangles_5.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	coordInfoVecs.nodes2Triangles_6.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	coordInfoVecs.nodes2Triangles_7.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	coordInfoVecs.nodes2Triangles_8.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	coordInfoVecs.nodes2Triangles_9.resize(mem_prealloc*generalParams.maxNodeCount,-INT_MAX);
	

	thrust::copy(coordInfoVecs.nodes2Triangles_1.begin(), coordInfoVecs.nodes2Triangles_1.end(), hostSetInfoVecs.nodes2Triangles_1.begin() );
	thrust::copy(coordInfoVecs.nodes2Triangles_2.begin(), coordInfoVecs.nodes2Triangles_2.end(), hostSetInfoVecs.nodes2Triangles_2.begin() );
	thrust::copy(coordInfoVecs.nodes2Triangles_3.begin(), coordInfoVecs.nodes2Triangles_3.end(), hostSetInfoVecs.nodes2Triangles_3.begin() );
	thrust::copy(coordInfoVecs.nodes2Triangles_4.begin(), coordInfoVecs.nodes2Triangles_4.end(), hostSetInfoVecs.nodes2Triangles_4.begin() );
	thrust::copy(coordInfoVecs.nodes2Triangles_5.begin(), coordInfoVecs.nodes2Triangles_5.end(), hostSetInfoVecs.nodes2Triangles_5.begin() );
	thrust::copy(coordInfoVecs.nodes2Triangles_6.begin(), coordInfoVecs.nodes2Triangles_6.end(), hostSetInfoVecs.nodes2Triangles_6.begin() );
	thrust::copy(coordInfoVecs.nodes2Triangles_7.begin(), coordInfoVecs.nodes2Triangles_7.end(), hostSetInfoVecs.nodes2Triangles_7.begin() );
	thrust::copy(coordInfoVecs.nodes2Triangles_8.begin(), coordInfoVecs.nodes2Triangles_8.end(), hostSetInfoVecs.nodes2Triangles_8.begin() );
	thrust::copy(coordInfoVecs.nodes2Triangles_9.begin(), coordInfoVecs.nodes2Triangles_9.end(), hostSetInfoVecs.nodes2Triangles_9.begin() );
	//thrust::copy(coordInfoVecs.nodes2Triangles_10.begin(), coordInfoVecs.nodes2Triangles_10.end(), hostInfoVecs.nodes2Triangles_10.begin() );
	//thrust::copy(coordInfoVecs.nodes2Triangles_11.begin(), coordInfoVecs.nodes2Triangles_11.end(), hostInfoVecs.nodes2Triangles_11.begin() );
	//thrust::copy(coordInfoVecs.nodes2Triangles_12.begin(), coordInfoVecs.nodes2Triangles_12.end(), hostInfoVecs.nodes2Triangles_12.begin() );

	//copy info to GPU
	std::cout<<"Copying"<<std::endl;
	thrust::copy(hostSetInfoVecs.isNodeFixed.begin(),hostSetInfoVecs.isNodeFixed.end(), coordInfoVecs.isNodeFixed.begin());
	
	std::cout<<"fixed_node_in_host: "<<std::endl;
	for (int k = 0; k < hostSetInfoVecs.isNodeFixed.size(); k++){
		//std::cout<<hostSetInfoVecs.isNodeFixed[k]<<std::endl;
	}
	std::cout<<"end_of_fixed_node_host_printout"<<std::endl;
	std::cout<<"fixed_node_in_device: "<<std::endl;
	for (int k = 0; k < coordInfoVecs.isNodeFixed.size(); k++){
		//std::cout<<coordInfoVecs.isNodeFixed[k]<<std::endl;
	}
	std::cout<<"end_of_fixed_node_device_printout"<<std::endl;
std::cout<<"size of host fixed "<< hostSetInfoVecs.isNodeFixed.size()<<std::endl;
std::cout<<"size of device fixed "<< coordInfoVecs.isNodeFixed.size()<<std::endl;

	/*for (int k = 0; k < coordInfoVecs.isNodeFixed.size(); k++){
		bool isFixedHost = hostSetInfoVecs.isNodeFixed[k];
		bool isFixedDevice = coordInfoVecs.isNodeFixed[k];
		if (isFixedDevice != isFixedHost){

			std::cout<<"pos "<< k << " dev val = " << coordInfoVecs.isNodeFixed[k]
				<< " host val = " <<  hostSetInfoVecs.isNodeFixed[k] <<std::endl;
		}
	}*/
	thrust::fill(coordInfoVecs.nodeForceX.begin(), coordInfoVecs.nodeForceX.end(), 0.0);
	thrust::fill(coordInfoVecs.nodeForceY.begin(), coordInfoVecs.nodeForceY.end(), 0.0);
	thrust::fill(coordInfoVecs.nodeForceZ.begin(), coordInfoVecs.nodeForceZ.end(), 0.0);

	thrust::fill(coordInfoVecs.prevNodeForceX.begin(), coordInfoVecs.prevNodeForceX.end(), 0.0);
	thrust::fill(coordInfoVecs.prevNodeForceY.begin(), coordInfoVecs.prevNodeForceY.end(), 0.0);
	thrust::fill(coordInfoVecs.prevNodeForceZ.begin(), coordInfoVecs.prevNodeForceZ.end(), 0.0);
	
	thrust::copy(hostSetInfoVecs.nodeLocX.begin(), hostSetInfoVecs.nodeLocX.end(), coordInfoVecs.prevNodeLocX.begin() );
	thrust::copy(hostSetInfoVecs.nodeLocY.begin(), hostSetInfoVecs.nodeLocY.end(), coordInfoVecs.prevNodeLocY.begin() );
	thrust::copy(hostSetInfoVecs.nodeLocZ.begin(), hostSetInfoVecs.nodeLocZ.end(), coordInfoVecs.prevNodeLocZ.begin() );
	
	thrust::copy(hostSetInfoVecs.nodeLocX.begin(), hostSetInfoVecs.nodeLocX.end(), coordInfoVecs.nodeLocX.begin() );
	thrust::copy(hostSetInfoVecs.nodeLocY.begin(), hostSetInfoVecs.nodeLocY.end(), coordInfoVecs.nodeLocY.begin() );
	thrust::copy(hostSetInfoVecs.nodeLocZ.begin(), hostSetInfoVecs.nodeLocZ.end(), coordInfoVecs.nodeLocZ.begin() );
	
	thrust::copy(hostSetInfoVecs.triangles2Nodes_1.begin(), hostSetInfoVecs.triangles2Nodes_1.end(), coordInfoVecs.triangles2Nodes_1.begin() );
	thrust::copy(hostSetInfoVecs.triangles2Nodes_2.begin(), hostSetInfoVecs.triangles2Nodes_2.end(), coordInfoVecs.triangles2Nodes_2.begin() );
	thrust::copy(hostSetInfoVecs.triangles2Nodes_3.begin(), hostSetInfoVecs.triangles2Nodes_3.end(), coordInfoVecs.triangles2Nodes_3.begin() );
	
	thrust::copy(hostSetInfoVecs.triangles2Edges_1.begin(), hostSetInfoVecs.triangles2Edges_1.end(), coordInfoVecs.triangles2Edges_1.begin() );
	thrust::copy(hostSetInfoVecs.triangles2Edges_2.begin(), hostSetInfoVecs.triangles2Edges_2.end(), coordInfoVecs.triangles2Edges_2.begin() );
	thrust::copy(hostSetInfoVecs.triangles2Edges_3.begin(), hostSetInfoVecs.triangles2Edges_3.end(), coordInfoVecs.triangles2Edges_3.begin() );

	thrust::copy(hostSetInfoVecs.edges2Nodes_1.begin(), hostSetInfoVecs.edges2Nodes_1.end(), coordInfoVecs.edges2Nodes_1.begin() );
	thrust::copy(hostSetInfoVecs.edges2Nodes_2.begin(), hostSetInfoVecs.edges2Nodes_2.end(), coordInfoVecs.edges2Nodes_2.begin() );
	
	thrust::copy(hostSetInfoVecs.edges2Triangles_1.begin(), hostSetInfoVecs.edges2Triangles_1.end(), coordInfoVecs.edges2Triangles_1.begin() );
	thrust::copy(hostSetInfoVecs.edges2Triangles_2.begin(), hostSetInfoVecs.edges2Triangles_2.end(), coordInfoVecs.edges2Triangles_2.begin() );

	thrust::copy(hostSetInfoVecs.nndata1.begin(), hostSetInfoVecs.nndata1.end(), coordInfoVecs.nndata1.begin() );
	thrust::copy(hostSetInfoVecs.nndata2.begin(), hostSetInfoVecs.nndata2.end(), coordInfoVecs.nndata2.begin() );
	thrust::copy(hostSetInfoVecs.nndata3.begin(), hostSetInfoVecs.nndata3.end(), coordInfoVecs.nndata3.begin() );
	thrust::copy(hostSetInfoVecs.nndata4.begin(), hostSetInfoVecs.nndata4.end(), coordInfoVecs.nndata4.begin() );
	thrust::copy(hostSetInfoVecs.nndata5.begin(), hostSetInfoVecs.nndata5.end(), coordInfoVecs.nndata5.begin() );
	thrust::copy(hostSetInfoVecs.nndata6.begin(), hostSetInfoVecs.nndata6.end(), coordInfoVecs.nndata6.begin() );
	thrust::copy(hostSetInfoVecs.nndata7.begin(), hostSetInfoVecs.nndata7.end(), coordInfoVecs.nndata7.begin() );
	thrust::copy(hostSetInfoVecs.nndata8.begin(), hostSetInfoVecs.nndata8.end(), coordInfoVecs.nndata8.begin() );
	thrust::copy(hostSetInfoVecs.nndata9.begin(), hostSetInfoVecs.nndata9.end(), coordInfoVecs.nndata9.begin() );
	//thrust::copy(hostSetInfoVecs.nndata10.begin(), hostSetInfoVecs.nndata10.end(), coordInfoVecs.nndata10.begin() );
	//thrust::copy(hostSetInfoVecs.nndata11.begin(), hostSetInfoVecs.nndata11.end(), coordInfoVecs.nndata11.begin() );
	//thrust::copy(hostSetInfoVecs.nndata12.begin(), hostSetInfoVecs.nndata12.end(), coordInfoVecs.nndata12.begin() );


 
	//allocate memory for other data structures.   

	//area triangle info vec
	//number of area springs is the number of triangles
	std::cout<<"Mem"<<std::endl;
	areaTriangleInfoVecs.tempNodeIdUnreduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceXUnreduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceYUnreduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceZUnreduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	
	areaTriangleInfoVecs.tempNodeIdReduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceXReduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceYReduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceZReduced.resize(mem_prealloc*areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);

	//beinding triangle info vec
	//num bending springs is the number of times each edge is between two triangles. 
	bendingTriangleInfoVecs.numBendingSprings = coordInfoVecs.num_edges;//coordInfoVecs.edges2Triangles_1.size();

	bendingTriangleInfoVecs.tempNodeIdUnreduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceXUnreduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceYUnreduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceZUnreduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	
	bendingTriangleInfoVecs.tempNodeIdReduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceXReduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceYReduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceZReduced.resize(mem_prealloc*bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);

	//linear springs
	
	linearSpringInfoVecs.tempNodeIdUnreduced.resize(mem_prealloc*linearSpringInfoVecs.factor*coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceXUnreduced.resize(mem_prealloc*linearSpringInfoVecs.factor*coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceYUnreduced.resize(mem_prealloc*linearSpringInfoVecs.factor*coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceZUnreduced.resize(mem_prealloc*linearSpringInfoVecs.factor*coordInfoVecs.num_edges);
	
	linearSpringInfoVecs.tempNodeIdReduced.resize(mem_prealloc*linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceXReduced.resize(mem_prealloc*linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceYReduced.resize(mem_prealloc*linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceZReduced.resize(mem_prealloc*linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
	
	linearSpringInfoVecs.edge_initial_length.clear();
	//linearSpringInfoVecs.edge_initial_length.resize(mem_prealloc*coordInfoVecs.num_edges,1.0);
	
	//thrust::copy(hostSetInfoVecs.edge_initial_length.begin(), hostSetInfoVecs.edge_initial_length.end(), linearSpringInfoVecs.edge_initial_length.begin() );

	//Resize the hostSetInfoVecs so that we can copy data back and forth between hostSetinfoVecs and coordInfoVecs without problem.
	hostSetInfoVecs.isNodeFixed.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	//hostSetInfoVecs.prevNodeLocX.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	//hostSetInfoVecs.prevNodeLocY.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	//hostSetInfoVecs.prevNodeLocZ.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());

	//hostSetInfoVecs.prevNodeForceX.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	//hostSetInfoVecs.prevNodeForceY.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	//hostSetInfoVecs.prevNodeForceZ.resize(mem_prealloc*hostSetInfoVecs.nodeLocX.size());
	
	hostSetInfoVecs.nodeLocX.resize(coordInfoVecs.nodeLocX.size());
	hostSetInfoVecs.nodeLocY.resize(coordInfoVecs.nodeLocX.size());
	hostSetInfoVecs.nodeLocZ.resize(coordInfoVecs.nodeLocX.size());
	std::cout<<"Host_nodeLocX size = "<<hostSetInfoVecs.nodeLocX.size()<<std::endl;

	hostSetInfoVecs.nodeForceX.resize(coordInfoVecs.nodeLocX.size());
	hostSetInfoVecs.nodeForceY.resize(coordInfoVecs.nodeLocX.size());
	hostSetInfoVecs.nodeForceZ.resize(coordInfoVecs.nodeLocX.size());
	std::cout<<"Host_nodeForceX size = "<<hostSetInfoVecs.nodeLocX.size()<<std::endl;

	hostSetInfoVecs.triangles2Nodes_1.resize( coordInfoVecs.triangles2Nodes_1.size() );
	hostSetInfoVecs.triangles2Nodes_2.resize( coordInfoVecs.triangles2Nodes_2.size() );
	hostSetInfoVecs.triangles2Nodes_3.resize( coordInfoVecs.triangles2Nodes_3.size() );
	std::cout<<"Host_triangles2Nodes size = "<<hostSetInfoVecs.triangles2Nodes_1.size()<<std::endl;
	
	hostSetInfoVecs.triangles2Edges_1.resize( coordInfoVecs.triangles2Edges_1.size() );
	hostSetInfoVecs.triangles2Edges_2.resize( coordInfoVecs.triangles2Edges_2.size() );
	hostSetInfoVecs.triangles2Edges_3.resize( coordInfoVecs.triangles2Edges_3.size() );
	std::cout<<"Host_triangles2Edges size = "<<hostSetInfoVecs.triangles2Edges_1.size()<<std::endl;

	hostSetInfoVecs.edges2Nodes_1.resize( coordInfoVecs.edges2Nodes_1.size() );
	hostSetInfoVecs.edges2Nodes_2.resize( coordInfoVecs.edges2Nodes_2.size() );
	std::cout<<"Host_edges2Nodes size = "<<hostSetInfoVecs.edges2Nodes_1.size()<<std::endl;
	
	hostSetInfoVecs.edges2Triangles_1.resize( coordInfoVecs.edges2Triangles_1.size() );
	hostSetInfoVecs.edges2Triangles_2.resize( coordInfoVecs.edges2Triangles_2.size() );
	std::cout<<"Host_edges2Triangles size = "<<hostSetInfoVecs.edges2Triangles_1.size()<<std::endl;

	hostSetInfoVecs.nndata1.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata2.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata3.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata4.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata5.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata6.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata7.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata8.resize( mem_prealloc*generalParams.maxNodeCount);
	hostSetInfoVecs.nndata9.resize( mem_prealloc*generalParams.maxNodeCount);
	//hostSetInfoVecs.nndata10.resize( mem_prealloc*generalParams.maxNodeCount);
	//hostSetInfoVecs.nndata11.resize( mem_prealloc*generalParams.maxNodeCount);
	//hostSetInfoVecs.nndata12.resize( mem_prealloc*generalParams.maxNodeCount);

	//std::cout<<"initial lengths: "<< linearSpringInfoVecs.edge_initial_length.size()<<std::endl;

	std::cout<<"System Ready"<<std::endl;

	//Generate LJ particle list. and set LJ particle midpoint.
	//double maxX_lj = *(thrust::max_element(coordInfoVecs.nodeLocX.begin(),coordInfoVecs.nodeLocX.end()));
	//double minX_lj = *(thrust::min_element(coordInfoVecs.nodeLocX.begin(),coordInfoVecs.nodeLocX.end()));
	//double maxY_lj = *(thrust::max_element(coordInfoVecs.nodeLocY.begin(),coordInfoVecs.nodeLocY.end()));
	//double minY_lj = *(thrust::min_element(coordInfoVecs.nodeLocY.begin(),coordInfoVecs.nodeLocY.end()));
	
	//ljInfoVecs.LJ_PosX = (maxX_lj + minX_lj)/2.0;
	//ljInfoVecs.LJ_PosY = (maxY_lj + minY_lj)/2.0;


	//currently unused
	/*thrust::host_vector<int> tempIds;
	for (int i = 0; i < hostSetInfoVecs.nodeLocX.size(); i++ ) {
		double xLoc = hostSetInfoVecs.nodeLocX[i];
		double yLoc = hostSetInfoVecs.nodeLocY[i];
		double zLoc = hostSetInfoVecs.nodeLocZ[i];
		
		double xDist = ljInfoVecs.LJ_PosX - xLoc;
		double yDist = ljInfoVecs.LJ_PosY - yLoc;
		double zDist = ljInfoVecs.LJ_PosZ - zLoc;

		double dist = std::sqrt(xDist*xDist + yDist*yDist + zDist*zDist);
		//just test all poitns for now. Optimize later.
		if (dist < ljInfoVecs.Rcutoff) {
			tempIds.push_back(i);
		}
	}
	ljInfoVecs.node_id_close.resize( tempIds.size() );
	thrust::copy(tempIds.begin(), tempIds.end(), ljInfoVecs.node_id_close.begin());
	std::cout<<"lj nodes: "<< ljInfoVecs.node_id_close.size() << std::endl;*/






	//last, set memory foor buckets.
	auxVecs.id_bucket.resize(generalParams.maxNodeCount);
	auxVecs.id_value.resize(generalParams.maxNodeCount);
	auxVecs.id_bucket_expanded.resize(27 * (generalParams.maxNodeCount));
	auxVecs.id_value_expanded.resize(27 *( generalParams.maxNodeCount ));
 


};


